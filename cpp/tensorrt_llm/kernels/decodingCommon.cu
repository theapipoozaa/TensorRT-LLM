#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "tensorrt_llm/common/cudaUtils.h"
#include "tensorrt_llm/common/reduceKernelUtils.cuh"
#include "tensorrt_llm/kernels/decodingCommon.h"
#include <stdio.h>

using namespace tensorrt_llm::common;

namespace tensorrt_llm
{
namespace kernels
{

__global__ void curandInitialize(hiprandState_t* state, const int* batchSlots, const int size, const uint64_t randomSeed)
{
    int const idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < size)
    {
        auto const batchSlot = batchSlots != nullptr ? batchSlots[idx] : idx;
        hiprand_init(randomSeed, 0, 0, &state[batchSlot]);
    }
}

void invokeCurandInitialize(
    hiprandState_t* state, const int* batchSlots, const size_t batchSize, const uint64_t randomSeed, hipStream_t stream)
{
    dim3 block(256);
    dim3 grid((int) (ceil(batchSize * 1.0 / 256)));
    curandInitialize<<<grid, block, 0, stream>>>(state, batchSlots, batchSize, randomSeed);
}

__global__ void curandBatchInitialize(
    hiprandState_t* states, const int* batchSlots, const int size, const uint64_t* randomSeeds)
{
    int const idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < size)
    {
        auto const batchSlot = batchSlots != nullptr ? batchSlots[idx] : idx;
        hiprand_init(randomSeeds[batchSlot], 0, 0, &states[batchSlot]);
    }
}

void invokeCurandBatchInitialize(hiprandState_t* states, const int* batchSlots, const size_t batchSize,
    const uint64_t* randomSeeds, hipStream_t stream)
{
    dim3 block(256);
    dim3 grid((int) (ceil(batchSize * 1.0 / 256)));
    curandBatchInitialize<<<grid, block, 0, stream>>>(states, batchSlots, batchSize, randomSeeds);
}

template <typename T>
__global__ void addBiasSoftMax(T* logits, T* probs, const T* bias, const int* endIds, const FinishedState* finished,
    const int* batchSlots, const int vocabSize, const int vocabSizePadded)
{
    auto const batchIdx = blockIdx.x;
    auto const batchSlot = batchSlots != nullptr ? batchSlots[batchIdx] : batchIdx;
    const FinishedState finishState = finished != nullptr ? finished[batchSlot] : FinishedState::empty();
    if (finishState.isSkipDecoding())
    {
        return;
    }

    bool finish = finishState.isFinished();
    int offset = batchIdx * vocabSizePadded;

    float maxVal = -1 * FLT_MAX;
    const bool IS_FP16 = std::is_same<T, half>::value;
    const T MAX_T_VAL = (IS_FP16) ? HALF_FLT_MAX : FLT_MAX;
    __shared__ float sMaxVal;
    __shared__ float sSumVal;

    for (int tid = threadIdx.x; tid < vocabSizePadded; tid += blockDim.x)
    {
        if (tid < vocabSize)
        {
            if (finish && endIds != nullptr)
            {
                logits[offset + tid] = (tid == endIds[batchSlot]) ? MAX_T_VAL : -MAX_T_VAL;
            }
            else
            {
                T bias_val = (bias != nullptr) ? bias[tid] : (T) 0.0f;
                logits[offset + tid] += bias_val;
            }
        }
        else
        {
            logits[offset + tid] = -MAX_T_VAL;
        }
        maxVal = max(maxVal, (float) logits[offset + tid]);
    }

    maxVal = blockReduceMax<float>((float) maxVal);
    if (threadIdx.x == 0)
    {
        sMaxVal = maxVal;
    }
    __syncthreads();

    float sumVal = 0.0f;
    for (int tid = threadIdx.x; tid < vocabSizePadded; tid += blockDim.x)
    {
        probs[offset + tid] = __expf((float) logits[offset + tid] - sMaxVal);
        sumVal += (float) probs[offset + tid];
    }

    sumVal = blockReduceSum<float>(sumVal);
    if (threadIdx.x == 0)
    {
        sSumVal = sumVal;
    }
    __syncthreads();

    for (int tid = threadIdx.x; tid < vocabSizePadded; tid += blockDim.x)
    {
        probs[offset + tid] = ((float) probs[offset + tid] / (sSumVal + 1e-6f));
    }
}

template <typename T>
void invokeAddBiasSoftMax(T* logits, T* probs, const T* bias, const int* endIds, const FinishedState* finished,
    const int* batchSlots, const int batchSize, const int vocabSize, const int vocabSizePadded, hipStream_t stream)
{
    dim3 grid(batchSize);
    auto const vocabRoundedToWarp = roundUp(vocabSize, 32);
    dim3 block(min(vocabRoundedToWarp, 1024));
    // vocabSize, e.g., 30000, 7000.... vocabSize is usually very big.
    addBiasSoftMax<<<grid, block, 0, stream>>>(
        logits, probs, bias, endIds, finished, batchSlots, vocabSize, vocabSizePadded);
}

template void invokeAddBiasSoftMax(float* logits, float* probs, const float* bias, const int* endIds,
    const FinishedState* finished, const int* batchSlots, const int m, const int nPadded, const int n,
    hipStream_t stream);

template void invokeAddBiasSoftMax(half* logits, half* probs, const half* bias, const int* endIds,
    const FinishedState* finished, const int* batchSlots, const int m, const int nPadded, const int n,
    hipStream_t stream);

template <typename T>
__global__ void scatterDecodingParamsKernel(T const* src, T* dst, int const* batchSlots, int batchSize)
{
    auto const batchIdx = blockIdx.x * blockDim.x + threadIdx.x;
    if (batchIdx >= batchSize)
    {
        return;
    }
    auto const batchSlot = batchSlots == nullptr ? batchIdx : batchSlots[batchIdx];
    dst[batchSlot] = src[batchIdx];
}

template <typename T>
void invokeScatterDecodingParams(T const* src, T* dst, int const* batchSlots, int batchSize, hipStream_t stream)
{
    constexpr int THREADS_PER_CTA = 256;
    dim3 grid(divUp(batchSize, THREADS_PER_CTA));
    scatterDecodingParamsKernel<<<grid, THREADS_PER_CTA, 0, stream>>>(src, dst, batchSlots, batchSize);
}

template void invokeScatterDecodingParams(
    float const* src, float* dst, int const* batchSlots, int batchSize, hipStream_t stream);
template void invokeScatterDecodingParams(
    uint32_t const* src, uint32_t* dst, int const* batchSlots, int batchSize, hipStream_t stream);
template void invokeScatterDecodingParams(
    int32_t const* src, int32_t* dst, int const* batchSlots, int batchSize, hipStream_t stream);
} // namespace kernels
} // namespace tensorrt_llm
