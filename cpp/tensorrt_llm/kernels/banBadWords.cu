#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022-2024, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "tensorrt_llm/common/cudaUtils.h"
#include "tensorrt_llm/kernels/banBadWords.h"

using namespace tensorrt_llm::common;

namespace tensorrt_llm
{
namespace kernels
{

template <typename T>
__global__ void ban_bad_words(T* logits, const int** output_ids_ptr, const int** parent_ids_ptr, const int* batch_slots,
    int batch_size, int beam_width, const int* bad_words, size_t bad_words_len, bool share_words, int vocab_size_padded,
    const int* sequence_lengths, const int max_seq_len)
{
    const int id = blockIdx.x * blockDim.x + threadIdx.x;
    const int batch_idx = blockIdx.y / beam_width;
    const int beam_idx = blockIdx.y % beam_width;
    auto const batch_slot = batch_slots != nullptr ? batch_slots[batch_idx] : batch_idx;
    auto const batch_beam_idx = batch_slot * beam_width + beam_idx;

    const int* base_bad_words = share_words ? bad_words : bad_words + batch_slot * 2 * bad_words_len;
    const int* base_bad_words_offsets = base_bad_words + bad_words_len;

    if (id >= bad_words_len || base_bad_words_offsets[id] < 0)
    {
        return;
    }

    const int item_end = base_bad_words_offsets[id];
    const int item_start = (id > 0) ? base_bad_words_offsets[id - 1] : 0;
    const int item_size = item_end - item_start;

    /* The single-token case unconditionally bans the token */
    bool should_ban = item_size == 1;
    const int current_step{sequence_lengths[batch_beam_idx]};
    /* Multi-token case and enough previously generated tokens to look for a match
     */
    if (item_size > 1 && current_step >= item_size - 1)
    {
        should_ban = true;
        int parent_id = beam_idx;
        const bool gather_beam = beam_width > 1;

        for (int token_idx = item_size - 2; token_idx >= 0; token_idx--)
        {
            const int previous_token
                = output_ids_ptr[batch_slot][parent_id * max_seq_len + current_step - (item_size - 1) + token_idx];

            if (previous_token != base_bad_words[item_start + token_idx])
            {
                should_ban = false;
                break;
            }
            if (gather_beam)
            {
                parent_id = parent_ids_ptr == nullptr
                    ? 0
                    : parent_ids_ptr[batch_slot][parent_id * max_seq_len + current_step - (item_size - 1) + token_idx];

                if (parent_id < 0 || parent_id >= beam_width)
                {
                    should_ban = false;
                    break;
                }
            }
        }
    }

    if (should_ban)
    {
        int banned_token = base_bad_words[item_end - 1];
        if (0 < banned_token && banned_token < vocab_size_padded)
        {
            logits[batch_slot * beam_width * vocab_size_padded + beam_idx * vocab_size_padded + banned_token]
                = static_cast<T>(-INFINITY);
        }
    }
}

template <typename T>
void invokeBanBadWords(T* logits, const int** output_ids_ptr, const int** parent_ids_ptr, const int* batch_slot,
    int batch_size, int local_batch_size, int beam_width, const int* bad_words, bool share_words, size_t bad_words_len,
    int vocab_size_padded, const int* sequence_lengths, int max_seq_len, hipStream_t stream)
{
    dim3 block, grid;
    constexpr size_t max_blocks{256};
    block.x = min(((bad_words_len + 32 - 1) / 32) * 32, max_blocks);
    grid.x = (bad_words_len + block.x - 1) / block.x;
    grid.y = local_batch_size * beam_width;

    ban_bad_words<<<grid, block, 0, stream>>>(logits, output_ids_ptr, parent_ids_ptr, batch_slot, batch_size,
        beam_width, bad_words, bad_words_len, share_words, vocab_size_padded, sequence_lengths, max_seq_len);
    sync_check_cuda_error();
}

template void invokeBanBadWords(half* logits, const int** output_ids_ptr, const int** parent_ids_ptr,
    const int* batch_slot, int batch_size, int local_batch_size, int beam_width, const int* bad_words, bool share_words,
    size_t bad_words_len, int vocab_size_padded, const int* sequence_lengths, int max_seq_len, hipStream_t stream);
#ifdef ENABLE_BF16
template void invokeBanBadWords(__hip_bfloat16* logits, const int** output_ids_ptr, const int** parent_ids_ptr,
    const int* batch_slot, int batch_size, int local_batch_size, int beam_width, const int* bad_words, bool share_words,
    size_t bad_words_len, int vocab_size_padded, const int* sequence_lengths, int max_seq_len, hipStream_t stream);
#endif
template void invokeBanBadWords(float* logits, const int** output_ids_ptr, const int** parent_ids_ptr,
    const int* batch_slot, int batch_size, int local_batch_size, int beam_width, const int* bad_words, bool share_words,
    size_t bad_words_len, int vocab_size_padded, const int* sequence_lengths, int max_seq_len, hipStream_t stream);

} // namespace kernels
} // namespace tensorrt_llm
