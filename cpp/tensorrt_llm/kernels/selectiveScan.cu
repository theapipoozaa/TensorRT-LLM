#include "hip/hip_runtime.h"
/*
 * Adapted from https://github.com/state-spaces/mamba/blob/main/csrc/selective_scan/selective_scan_fwd_kernel.cuh
 * Copyright (c) 2023, Tri Dao.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 * Not a contribution
 * Changes made by NVIDIA CORPORATION & AFFILIATES or otherwise documented as
 * NVIDIA-proprietary are not a contribution and subject to the following terms and conditions:
 * SPDX-FileCopyrightText: Copyright (c) 2022-2024 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: LicenseRef-NvidiaProprietary
 *
 * NVIDIA CORPORATION, its affiliates and licensors retain all intellectual
 * property and proprietary rights in and to this material, related
 * documentation and any modifications thereto. Any use, reproduction,
 * disclosure or distribution of this material and related documentation
 * without an express license agreement from NVIDIA CORPORATION or
 * its affiliates is strictly prohibited.
 */

#include <hip/hip_runtime_api.h>
#ifdef ENABLE_FP8
#include <hip/hip_fp8.h>
#endif

#include <cub/block/block_load.cuh>
#include <cub/block/block_scan.cuh>
#include <cub/block/block_store.cuh>

#include "selectiveScan.h"
#include "selectiveScanCommon.h"
#ifndef M_LOG2E
#define M_LOG2E 1.4426950408889634
#endif
namespace tensorrt_llm
{
namespace kernels
{

template <int kNThreads_, int kNItems_, int kNRows_, bool kIsEvenLen_, bool kIsVariableB_, bool kIsVariableC_,
    bool kHasZ_, typename input_t_, typename weight_t_>
struct Selective_Scan_fwd_kernel_traits
{
    static_assert(kNItems_ % 4 == 0);
    using input_t = input_t_;
    using weight_t = weight_t_;
    static constexpr int kNThreads = kNThreads_;
    // Setting MinBlocksPerMP to be 3 (instead of 2) for 128 threads improves occupancy.
    static constexpr int kMinBlocks = kNThreads < 128 ? 5 : 3;
    static constexpr int kNItems = kNItems_;
    static constexpr int kNRows = kNRows_;
    static constexpr int kNBytes = sizeof(input_t);
    static_assert(kNBytes == 2 || kNBytes == 4);
    static constexpr int kNElts = kNBytes == 4 ? 4 : std::min(8, kNItems);
    static_assert(kNItems % kNElts == 0);
    static constexpr int kNLoads = kNItems / kNElts;
    static constexpr bool kIsEvenLen = kIsEvenLen_;
    static constexpr bool kIsVariableB = kIsVariableB_;
    static constexpr bool kIsVariableC = kIsVariableC_;
    static constexpr bool kHasZ = kHasZ_;

    static constexpr bool kDirectIO = kIsEvenLen && kNLoads == 1;

    using vec_t = typename BytesToType<kNBytes * kNElts>::Type;
    using scan_t = float2;
    using scan_t_s = float;
    using BlockLoadT = hipcub::BlockLoad<input_t, kNThreads, kNItems, hipcub::BLOCK_LOAD_WARP_TRANSPOSE>;
    using BlockLoadVecT = hipcub::BlockLoad<vec_t, kNThreads, kNLoads,
        !kDirectIO ? hipcub::BLOCK_LOAD_WARP_TRANSPOSE : hipcub::BLOCK_LOAD_DIRECT>;
    using BlockLoadWeightT = hipcub::BlockLoad<input_t, kNThreads, kNItems, hipcub::BLOCK_LOAD_WARP_TRANSPOSE>;
    using BlockLoadWeightVecT = hipcub::BlockLoad<vec_t, kNThreads, kNLoads,
        !kDirectIO ? hipcub::BLOCK_LOAD_WARP_TRANSPOSE : hipcub::BLOCK_LOAD_DIRECT>;
    using BlockStoreT = hipcub::BlockStore<input_t, kNThreads, kNItems, hipcub::BLOCK_STORE_WARP_TRANSPOSE>;
    using BlockStoreVecT = hipcub::BlockStore<vec_t, kNThreads, kNLoads,
        !kDirectIO ? hipcub::BLOCK_STORE_WARP_TRANSPOSE : hipcub::BLOCK_STORE_DIRECT>;
    // using BlockScanT = hipcub::BlockScan<scan_t, kNThreads, hipcub::BLOCK_SCAN_RAKING_MEMOIZE>;
    // using BlockScanT = hipcub::BlockScan<scan_t, kNThreads, hipcub::BLOCK_SCAN_RAKING>;
    using BlockScanT = hipcub::BlockScan<scan_t, kNThreads, hipcub::BLOCK_SCAN_WARP_SCANS>;
    static constexpr int kSmemIOSize
        = std::max({sizeof(typename BlockLoadT::TempStorage), sizeof(typename BlockLoadVecT::TempStorage),
            (int(kIsVariableB) + int(kIsVariableC)) * sizeof(typename BlockLoadWeightT::TempStorage),
            (int(kIsVariableB) + int(kIsVariableC)) * sizeof(typename BlockLoadWeightVecT::TempStorage),
            sizeof(typename BlockStoreT::TempStorage), sizeof(typename BlockStoreVecT::TempStorage)});
    static constexpr int kSmemSize = kSmemIOSize + sizeof(typename BlockScanT::TempStorage);
};

template <typename Ktraits>
__global__ __launch_bounds__(Ktraits::kNThreads, Ktraits::kMinBlocks) void selective_scan_fwd_kernel(
    SSMParamsBase params)
{
    constexpr bool kIsVariableB = Ktraits::kIsVariableB;
    constexpr bool kIsVariableC = Ktraits::kIsVariableC;
    constexpr bool kHasZ = Ktraits::kHasZ;
    constexpr int kNThreads = Ktraits::kNThreads;
    constexpr int kNItems = Ktraits::kNItems;
    constexpr int kNRows = Ktraits::kNRows;
    constexpr bool kDirectIO = Ktraits::kDirectIO;
    using input_t = typename Ktraits::input_t;
    using weight_t = typename Ktraits::weight_t;
    using scan_t = typename Ktraits::scan_t;
    using scan_t_s = typename Ktraits::scan_t_s;

    // Shared memory.
    extern __shared__ char smem_[];
    // cast to lvalue reference of expected type
    // char *smem_loadstorescan = smem_ + 2 * MAX_DSTATE * sizeof(weight_t);
    // auto& smem_load = reinterpret_cast<typename BlockLoadT::TempStorage&>(smem_ + 2 * MAX_DSTATE * sizeof(weight_t));
    // auto& smem_load = reinterpret_cast<typename BlockLoadT::TempStorage&>(smem_loadstorescan);
    auto& smem_load = reinterpret_cast<typename Ktraits::BlockLoadT::TempStorage&>(smem_);
    auto& smem_load_weight = reinterpret_cast<typename Ktraits::BlockLoadWeightT::TempStorage&>(smem_);
    auto& smem_load_weight1 = *reinterpret_cast<typename Ktraits::BlockLoadWeightT::TempStorage*>(
        smem_ + sizeof(typename Ktraits::BlockLoadWeightT::TempStorage));
    auto& smem_store = reinterpret_cast<typename Ktraits::BlockStoreT::TempStorage&>(smem_);
    auto& smem_scan = *reinterpret_cast<typename Ktraits::BlockScanT::TempStorage*>(smem_ + Ktraits::kSmemIOSize);
    // weight_t *smem_a = reinterpret_cast<weight_t *>(smem_ + smem_loadstorescan_size);
    // weight_t *smem_bc = reinterpret_cast<weight_t *>(smem_a + MAX_DSTATE);
    scan_t* smem_running_prefix = reinterpret_cast<scan_t*>(smem_ + Ktraits::kSmemSize);

    const int batch_id = blockIdx.x;
    const int dim_id = blockIdx.y;
    const int group_id = dim_id / (params.dim_ngroups_ratio);
    input_t* u = reinterpret_cast<input_t*>(params.u_ptr) + batch_id * params.u_batch_stride
        + dim_id * kNRows * params.u_d_stride;
    input_t* delta = reinterpret_cast<input_t*>(params.delta_ptr) + batch_id * params.delta_batch_stride
        + dim_id * kNRows * params.delta_d_stride;
    weight_t* A = reinterpret_cast<weight_t*>(params.A_ptr) + dim_id * kNRows * params.A_d_stride;
    weight_t* B = reinterpret_cast<weight_t*>(params.B_ptr) + dim_id * kNRows * params.B_d_stride;
    input_t* Bvar = reinterpret_cast<input_t*>(params.B_ptr) + batch_id * params.B_batch_stride
        + group_id * params.B_group_stride;
    weight_t* C = reinterpret_cast<weight_t*>(params.C_ptr) + dim_id * kNRows * params.C_d_stride;
    input_t* Cvar = reinterpret_cast<input_t*>(params.C_ptr) + batch_id * params.C_batch_stride
        + group_id * params.C_group_stride;
    scan_t_s* x = reinterpret_cast<scan_t_s*>(params.x_ptr) + (batch_id * params.dim + dim_id * kNRows) * params.dstate;

    float D_val[kNRows] = {0};
    if (params.D_ptr != nullptr)
    {
#pragma unroll
        for (int r = 0; r < kNRows; ++r)
        {
            D_val[r] = reinterpret_cast<float*>(params.D_ptr)[dim_id * kNRows + r];
        }
    }
    float delta_bias[kNRows] = {0};
    if (params.delta_bias_ptr != nullptr)
    {
#pragma unroll
        for (int r = 0; r < kNRows; ++r)
        {
            delta_bias[r] = reinterpret_cast<float*>(params.delta_bias_ptr)[dim_id * kNRows + r];
        }
    }

    // for (int state_idx = threadIdx.x; state_idx < params.dstate; state_idx += blockDim.x) {
    //     smem_a[state_idx] = A[state_idx * params.A_dstate_stride];
    //     smem_bc[state_idx] = B[state_idx * params.B_dstate_stride] * C[state_idx * params.C_dstate_stride];
    // }

    constexpr int kChunkSize = kNThreads * kNItems;
    for (int chunk = 0; chunk < params.n_chunks; ++chunk)
    {
        input_t u_vals[kNRows][kNItems], delta_vals_load[kNRows][kNItems];
        __syncthreads();
#pragma unroll
        for (int r = 0; r < kNRows; ++r)
        {
            if constexpr (!kDirectIO)
            {
                if (r > 0)
                {
                    __syncthreads();
                }
            }
            load_input<Ktraits>(u + r * params.u_d_stride, u_vals[r], smem_load, params.seqlen - chunk * kChunkSize);
            if constexpr (!kDirectIO)
            {
                __syncthreads();
            }
            load_input<Ktraits>(
                delta + r * params.delta_d_stride, delta_vals_load[r], smem_load, params.seqlen - chunk * kChunkSize);
        }
        u += kChunkSize;
        delta += kChunkSize;

        float delta_vals[kNRows][kNItems], delta_u_vals[kNRows][kNItems], out_vals[kNRows][kNItems];
#pragma unroll
        for (int r = 0; r < kNRows; ++r)
        {
#pragma unroll
            for (int i = 0; i < kNItems; ++i)
            {
                float u_val = float(u_vals[r][i]);
                delta_vals[r][i] = float(delta_vals_load[r][i]) + delta_bias[r];
                if (params.delta_softplus)
                {
                    delta_vals[r][i] = delta_vals[r][i] <= 20.f ? log1pf(expf(delta_vals[r][i])) : delta_vals[r][i];
                }
                delta_u_vals[r][i] = delta_vals[r][i] * u_val;
                out_vals[r][i] = D_val[r] * u_val;
            }
        }

        __syncthreads();
        for (int state_idx = 0; state_idx < params.dstate; ++state_idx)
        {
            weight_t A_val[kNRows];
#pragma unroll
            for (int r = 0; r < kNRows; ++r)
            {
                A_val[r] = A[state_idx * params.A_dstate_stride + r * params.A_d_stride];
                // Multiply the real part of A with LOG2E so we can use exp2f instead of expf.
                constexpr float kLog2e = M_LOG2E;
                A_val[r] *= kLog2e;
            }
            // This variable holds B * C if both B and C are constant across seqlen. If only B varies
            // across seqlen, this holds C. If only C varies across seqlen, this holds B.
            // If both B and C vary, this is unused.
            weight_t BC_val[kNRows];
            weight_t B_vals[kNItems], C_vals[kNItems];
            if constexpr (kIsVariableB)
            {
                load_weight<Ktraits>(Bvar + state_idx * params.B_dstate_stride, B_vals, smem_load_weight,
                    params.seqlen - chunk * kChunkSize);
                if constexpr (!kIsVariableC)
                {
#pragma unroll
                    for (int r = 0; r < kNRows; ++r)
                    {
                        BC_val[r] = C[state_idx * params.C_dstate_stride + r * params.C_d_stride];
                    }
                }
            }
            if constexpr (kIsVariableC)
            {
                auto& smem_load_weight_C = !kIsVariableB ? smem_load_weight : smem_load_weight1;
                load_weight<Ktraits>(Cvar + state_idx * params.C_dstate_stride, C_vals, smem_load_weight_C,
                    params.seqlen - chunk * kChunkSize);
                if constexpr (!kIsVariableB)
                {
#pragma unroll
                    for (int r = 0; r < kNRows; ++r)
                    {
                        BC_val[r] = B[state_idx * params.B_dstate_stride + r * params.B_d_stride];
                    }
                }
            }
            if constexpr (!kIsVariableB && !kIsVariableC)
            {
#pragma unroll
                for (int r = 0; r < kNRows; ++r)
                {
                    BC_val[r] = B[state_idx * params.B_dstate_stride + r * params.B_d_stride]
                        * C[state_idx * params.C_dstate_stride + r * params.C_d_stride];
                }
            }

#pragma unroll
            for (int r = 0; r < kNRows; ++r)
            {
                if (r > 0)
                {
                    __syncthreads();
                } // Scan could be using the same smem
                scan_t thread_data[kNItems];
#pragma unroll
                for (int i = 0; i < kNItems; ++i)
                {
                    thread_data[i] = make_float2(exp2f(delta_vals[r][i] * A_val[r]),
                        !kIsVariableB ? delta_u_vals[r][i] : B_vals[i] * delta_u_vals[r][i]);
                    if constexpr (!Ktraits::kIsEvenLen)
                    { // So that the last state is correct
                        if (threadIdx.x * kNItems + i >= params.seqlen - chunk * kChunkSize)
                        {
                            thread_data[i] = make_float2(1.f, 0.f);
                        }
                    }
                }
                // Initialize running total
                scan_t running_prefix;
                // If we use WARP_SCAN then all lane 0 of all warps (not just thread 0) needs to read
                running_prefix = chunk > 0 && threadIdx.x % 32 == 0 ? smem_running_prefix[state_idx + r * MAX_DSTATE]
                                                                    : make_float2(1.f, 0.f);
                // running_prefix = chunk > 0 && threadIdx.x == 0 ? smem_running_prefix[state_idx] :
                // make_float2(1.f, 0.f);
                SSMScanPrefixCallbackOp<weight_t> prefix_op(running_prefix);
                Ktraits::BlockScanT(smem_scan).InclusiveScan(
                    thread_data, thread_data, SSMScanOp<weight_t>(), prefix_op);
                // There's a syncthreads in the scan op, so we don't need to sync here.
                // Unless there's only 1 warp, but then it's the same thread (0) reading and writing.
                if (threadIdx.x == 0)
                {
                    smem_running_prefix[state_idx] = prefix_op.running_prefix;
                    if (chunk == params.n_chunks - 1)
                    {
                        x[r * params.dstate + state_idx] = prefix_op.running_prefix.y;
                    }
                }
#pragma unroll
                for (int i = 0; i < kNItems; ++i)
                {
                    const weight_t C_val
                        = !kIsVariableC ? BC_val[r] : (!kIsVariableB ? BC_val[r] * C_vals[i] : C_vals[i]);
                    out_vals[r][i] += thread_data[i].y * C_val;
                }
            }
        }

        input_t* out = reinterpret_cast<input_t*>(params.out_ptr) + batch_id * params.out_batch_stride
            + dim_id * kNRows * params.out_d_stride + chunk * kChunkSize;
        if constexpr (kHasZ)
        {
            input_t* z = reinterpret_cast<input_t*>(params.z_ptr) + batch_id * params.z_batch_stride
                + dim_id * kNRows * params.z_d_stride + chunk * kChunkSize;
#pragma unroll
            for (int r = 0; r < kNRows; ++r)
            {
                input_t z_vals[kNItems];
                __syncthreads();
                load_input<Ktraits>(z + r * params.z_d_stride, z_vals, smem_load, params.seqlen - chunk * kChunkSize);
#pragma unroll
                for (int i = 0; i < kNItems; ++i)
                {
                    float z_val = z_vals[i];
                    out_vals[r][i] *= z_val / (1 + expf(-z_val));
                }
                __syncthreads();
                store_output<Ktraits>(
                    out + r * params.out_d_stride, out_vals[r], smem_store, params.seqlen - chunk * kChunkSize);
            }
        }
        else
        {
            __syncthreads();
#pragma unroll
            for (int r = 0; r < kNRows; ++r)
            {
                if constexpr (!kDirectIO)
                {
                    if (r > 0)
                    {
                        __syncthreads();
                    }
                }
                store_output<Ktraits>(
                    out + r * params.out_d_stride, out_vals[r], smem_store, params.seqlen - chunk * kChunkSize);
            }
        }

        Bvar += kChunkSize;
        Cvar += kChunkSize;
    }
}

template <int kNThreads, int kNItems, typename input_t, typename weight_t>
void selective_scan_fwd_launch(SSMParamsBase& params, hipStream_t stream)
{
    // Only kNRows == 1 is tested for now, which ofc doesn't differ from previously when we had each block
    // processing 1 row.
    constexpr int kNRows = 1;
    BOOL_SWITCH(params.seqlen % (kNThreads * kNItems) == 0, kIsEvenLen,
        [&]
        {
            BOOL_SWITCH(params.is_variable_B, kIsVariableB,
                [&]
                {
                    BOOL_SWITCH(params.is_variable_C, kIsVariableC,
                        [&]
                        {
                            BOOL_SWITCH(params.z_ptr != nullptr, kHasZ,
                                [&]
                                {
                                    using Ktraits = Selective_Scan_fwd_kernel_traits<kNThreads, kNItems, kNRows,
                                        kIsEvenLen, kIsVariableB, kIsVariableC, kHasZ, input_t, weight_t>;
                                    // constexpr int kSmemSize = Ktraits::kSmemSize;
                                    constexpr int kSmemSize
                                        = Ktraits::kSmemSize + kNRows * MAX_DSTATE * sizeof(typename Ktraits::scan_t);
                                    // printf("smem_size = %d\n", kSmemSize);
                                    dim3 grid(params.batch, params.dim / kNRows);
                                    auto kernel = &selective_scan_fwd_kernel<Ktraits>;
                                    if (kSmemSize >= 48 * 1024)
                                    {
                                        TLLM_CUDA_CHECK(hipFuncSetAttribute(reinterpret_cast<const void*>(
                                            kernel), hipFuncAttributeMaxDynamicSharedMemorySize, kSmemSize));
                                    }
                                    kernel<<<grid, Ktraits::kNThreads, kSmemSize, stream>>>(params);
                                });
                        });
                });
        });
}

template <typename input_t, typename weight_t>
void invokeSelectiveScan(SSMParamsBase& params, hipStream_t stream)
{
    if (params.seqlen <= 128)
    {
        selective_scan_fwd_launch<32, 4, input_t, weight_t>(params, stream);
    }
    else if (params.seqlen <= 256)
    {
        selective_scan_fwd_launch<32, 8, input_t, weight_t>(params, stream);
    }
    else if (params.seqlen <= 512)
    {
        selective_scan_fwd_launch<32, 16, input_t, weight_t>(params, stream);
    }
    else if (params.seqlen <= 1024)
    {
        selective_scan_fwd_launch<64, 16, input_t, weight_t>(params, stream);
    }
    else
    {
        selective_scan_fwd_launch<128, 16, input_t, weight_t>(params, stream);
    }
}

#define INSTANTIATE_SELECTIVE_SCAN_DATA_TYPE(input_t, weight_t)                                                        \
    template void invokeSelectiveScan<input_t, weight_t>(SSMParamsBase & params, hipStream_t stream);

INSTANTIATE_SELECTIVE_SCAN_DATA_TYPE(float, float);
INSTANTIATE_SELECTIVE_SCAN_DATA_TYPE(half, float);
#ifdef ENABLE_BF16
INSTANTIATE_SELECTIVE_SCAN_DATA_TYPE(__hip_bfloat16, float);
#endif
#undef INSTANTIATE_SELECTIVE_SCAN_DATA_TYPE

////////////////////////////////////////////////////////////////////////////////////////////////////

template <typename input_t, typename weight_t, bool dt_softplus, bool has_dt_bias, bool has_d, bool has_z>
__global__ void selectiveScanUpdate(SSMParamsBase params)
{
    // Shared memory.
    extern __shared__ char smem_[];

    input_t* smem_b = reinterpret_cast<input_t*>(smem_);
    input_t* smem_c = reinterpret_cast<input_t*>(smem_ + sizeof(input_t) * params.dstate);

    const int batch_id = blockIdx.x;
    const int dim_id = blockIdx.y * blockDim.x + threadIdx.x;

    const input_t x = reinterpret_cast<const input_t*>(params.u_ptr)[batch_id * params.u_batch_stride + dim_id];
    const weight_t* A = reinterpret_cast<const weight_t*>(params.A_ptr) + dim_id * params.A_d_stride;
    const input_t* B = reinterpret_cast<const input_t*>(params.B_ptr) + batch_id * params.B_batch_stride;
    const input_t* C = reinterpret_cast<const input_t*>(params.C_ptr) + batch_id * params.C_batch_stride;
    const float* D_ptr = reinterpret_cast<const float*>(params.D_ptr);
    const input_t* z_ptr = reinterpret_cast<const input_t*>(params.z_ptr);
    weight_t* state = reinterpret_cast<weight_t*>(params.x_ptr) + batch_id * params.state_batch_stride
        + dim_id * params.state_d_stride;
    const input_t dt
        = reinterpret_cast<const input_t*>(params.delta_ptr)[batch_id * params.delta_batch_stride + dim_id];
    const float* dt_bias_ptr = reinterpret_cast<const float*>(params.delta_bias_ptr);
    input_t* out = reinterpret_cast<input_t*>(params.out_ptr) + batch_id * params.out_batch_stride;
    float out_tmp = 0.0f;

    // get delta bias
    float dt_bias = 0.0f;
    if (has_dt_bias)
    {
        dt_bias = dt_bias_ptr[dim_id];
    }

    // get D
    float D = 0.0f;
    if (has_d)
    {
        D = D_ptr[dim_id];
    }

    // dt = softplus(dt + dt_bias)
    float dt_val = float(dt) + dt_bias;
    if (dt_softplus)
    {
        dt_val = dt_val <= 20.f ? log1pf(expf(dt_val)) : dt_val;
    }

    out_tmp = D * float(x);

    // read B, C
    if (threadIdx.x == 0)
    {
#pragma unroll
        for (int i = 0; i < params.dstate; ++i)
        {
            smem_b[i] = B[i];
            smem_c[i] = C[i];
        }
    }
    __syncthreads();

    for (int state_idx = 0; state_idx < params.dstate; ++state_idx)
    {
        // read A
        weight_t A_val = A[state_idx];

        // Multiply the real part of A with LOG2E so we can use exp2f instead of expf.
        constexpr float kLog2e = M_LOG2E;
        A_val *= kLog2e;

        // dtA = exp(dt * A), dtB = dt * B
        float dt_A = exp2f(dt_val * A_val);
        float dt_B = dt_val * float(smem_b[state_idx]);

        // update state
        float state_new = float(state[state_idx]) * dt_A + float(x) * dt_B;
        state[state_idx] = weight_t(state_new);

        // y = C * state + D * x
        out_tmp += state_new * float(smem_c[state_idx]);
    }

    // y = y * silu(z)
    if (has_z)
    {
        float z = z_ptr[batch_id * params.z_batch_stride + dim_id];
        out_tmp *= z / (1 + expf(-z));
    }

    // save out
    out[dim_id] = input_t(out_tmp);
}

template <typename input_t, typename weight_t>
void invokeSelectiveScanUpdate(SSMParamsBase& params, hipStream_t stream)
{
    const int kNThreads = 32;
    dim3 block(kNThreads);
    dim3 grid(params.batch, (params.dim + kNThreads - 1) / kNThreads);
    // only save B and C to shared mem for reuse
    size_t smem_size = params.dstate * sizeof(input_t) * 2;

    BOOL_SWITCH(params.delta_softplus, kDtSoftplus,
        [&]
        {
            BOOL_SWITCH(params.delta_bias_ptr != nullptr, kHasDtBias,
                [&]
                {
                    BOOL_SWITCH(params.D_ptr != nullptr, kHasD,
                        [&]
                        {
                            BOOL_SWITCH(params.z_ptr != nullptr, kHasZ,
                                [&]
                                {
                                    selectiveScanUpdate<input_t, weight_t, kDtSoftplus, kHasDtBias, kHasD, kHasZ>
                                        <<<grid, block, smem_size, stream>>>(params);
                                });
                        });
                });
        });
}

#define INSTANTIATE_SELECTIVE_SCAN_UPDATE_DATA_TYPE(input_t, weight_t)                                                 \
    template void invokeSelectiveScanUpdate<input_t, weight_t>(SSMParamsBase & params, hipStream_t stream)

INSTANTIATE_SELECTIVE_SCAN_UPDATE_DATA_TYPE(float, float);
INSTANTIATE_SELECTIVE_SCAN_UPDATE_DATA_TYPE(half, float);
#ifdef ENABLE_BF16
INSTANTIATE_SELECTIVE_SCAN_UPDATE_DATA_TYPE(__hip_bfloat16, float);
#endif
#undef INSTANTIATE_SELECTIVE_SCAN_UPDATE_DATA_TYPE

} // namespace kernels
} // namespace tensorrt_llm
