#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022-2024, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "tensorrt_llm/common/cudaUtils.h"
#include "tensorrt_llm/common/reduceKernelUtils.cuh"
#include "tensorrt_llm/kernels/stopCriteriaKernels.h"

using namespace tensorrt_llm::common;

namespace tensorrt_llm
{
namespace kernels
{
__global__ void stopWordsCriterion(const int** outputIds, const int** parentIds, const int* stopWords,
    FinishedState* finished, const int* sequenceLengths, const int* batchSlots, size_t stopWordsLen, int batchSize,
    int beamWidth, int maxSeqLen)
{
    int const id = blockIdx.x * blockDim.x + threadIdx.x;
    int const batchIdx = blockIdx.y / beamWidth;
    int const beamIdx = blockIdx.y % beamWidth;
    auto const batchSlot = batchSlots != nullptr ? batchSlots[batchIdx] : batchIdx;
    auto const batchBeamIdx = batchSlot * beamWidth + beamIdx;

    int const* baseStopWords = stopWords + batchSlot * 2 * stopWordsLen;
    int const* baseOffsets = baseStopWords + stopWordsLen;

    if (id >= stopWordsLen || baseOffsets[id] < 0)
    {
        return;
    }

    int const itemEnd = baseOffsets[id];
    int const itemStart = (id > 0) ? baseOffsets[id - 1] : 0;
    int const itemSize = itemEnd - itemStart;

    // The single-token case unconditionally bans the token
    bool shouldStop = false;

    // Need to minus 1 because the sequenceLengths is updated in this step
    int const currentStep = sequenceLengths[batchBeamIdx] - 1;
    // Enough previously generated tokens to look for a match
    if (currentStep + 1 >= itemSize)
    {
        shouldStop = true;
        int parentId = beamIdx;
        bool const gatherBeam = beamWidth > 1;

        for (int tokenIdx = itemSize - 1; tokenIdx >= 0; tokenIdx--)
        {
            int const previousToken
                = outputIds[batchSlot][parentId * maxSeqLen + currentStep - (itemSize - 1) + tokenIdx];
            if (previousToken != baseStopWords[itemStart + tokenIdx])
            {
                shouldStop = false;
                break;
            }
            if (gatherBeam)
            {
                parentId = parentIds == nullptr
                    ? 0
                    : parentIds[batchSlot][parentId * maxSeqLen + currentStep - (itemSize - 1) + tokenIdx];

                if (parentId < 0 || parentId >= beamWidth)
                {
                    shouldStop = false;
                    break;
                }
            }
        }
    }

    if (shouldStop)
    {
        finished[batchSlot * beamWidth + beamIdx].setFinishedStopWords();
    }
}

void invokeStopWordsCriterion(const int** outputIds, const int** parentIds, const int* stopWords,
    FinishedState* finished, const int* sequenceLengths, const int* batchSlots, size_t stopWordsLen, int batchSize,
    int beamWidth, int maxSeqLen, hipStream_t stream)
{
    // Check if we have sampled a word from the stopWords list. If so, stop the sequence.
    dim3 block, grid;
    constexpr size_t maxBlockSize{256};
    block.x = min(((stopWordsLen + 32 - 1) / 32) * 32, maxBlockSize);
    grid.x = (stopWordsLen + block.x - 1) / block.x;
    grid.y = batchSize * beamWidth;

    stopWordsCriterion<<<grid, block, 0, stream>>>(outputIds, parentIds, stopWords, finished, sequenceLengths,
        batchSlots, stopWordsLen, batchSize, beamWidth, maxSeqLen);
    sync_check_cuda_error();
}

__global__ void lengthCriterion(FinishedState* finished, int* finishedSum, const uint32_t* sequenceLimitLength,
    const int* sequenceLengths, const int* batchSlots, int batchSize, int beamWidth)
{
    int threadFinishedCount = 0;
    for (int index = threadIdx.x; index < batchSize * beamWidth; index += blockDim.x)
    {
        int const batchIdx = index / beamWidth;
        int const beamIdx = index % beamWidth;
        auto const batchSlot = batchSlots != nullptr ? batchSlots[batchIdx] : batchIdx;
        auto const batchSlotBeamWidthIdx = batchSlot * beamWidth + beamIdx;

        auto finishState = finished[batchSlotBeamWidthIdx];

        if (sequenceLengths[batchSlotBeamWidthIdx] >= sequenceLimitLength[batchSlot])
        {
            finishState.setFinishedMaxLength();
        }
        threadFinishedCount += finishState.isFinished() ? 1 : 0;
        finished[batchSlotBeamWidthIdx] = finishState;
    }

    if (finishedSum)
    {
        int blockFinishedCount = 0;
        if (blockDim.x <= 32)
        {
            blockFinishedCount = warpReduceSum(threadFinishedCount);
        }
        else
        {
            blockFinishedCount = blockReduceSum(threadFinishedCount);
        }
        __syncthreads();

        if (threadIdx.x == 0)
        {
            finishedSum[0] = blockFinishedCount;
        }
    }
}

void invokeLengthCriterion(FinishedState* finished, int* finishedSum, const uint32_t* sequenceLimitLength,
    const int* sequenceLengths, const int* batchSlots, int batchSize, int beamWidth, hipStream_t stream)
{
    // Check if we have attained the sequence length limit. If so, stop the
    // sequence. In addition, check if all sequences are stopped and return the
    // result in shouldStop
    dim3 block{min(512, uint32_t(batchSize * beamWidth))};
    dim3 grid{1};

    lengthCriterion<<<grid, block, 0, stream>>>(
        finished, finishedSum, sequenceLimitLength, sequenceLengths, batchSlots, batchSize, beamWidth);
    sync_check_cuda_error();
}

} // namespace kernels
} // namespace tensorrt_llm
