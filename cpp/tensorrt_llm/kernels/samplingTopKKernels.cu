#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2023, NVIDIA CORPORATION.  All rights reserved.
 * Copyright (c) 2021, NAVER Corp.  Authored by CLOVA.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <stdexcept>
#ifndef CUDART_VERSION
#error CUDART_VERSION Undefined!
#elif (CUDART_VERSION >= 11050)
#include <hipcub/hipcub.hpp>
#else
#include "3rdparty/hipcub/hipcub.hpp"
#endif

#include "tensorrt_llm/common/logger.h"
#include "tensorrt_llm/common/reduceKernelUtils.cuh"
#include "tensorrt_llm/common/stringUtils.h"
#include "tensorrt_llm/kernels/samplingTopKKernels.h"

using namespace tensorrt_llm::common;

namespace tensorrt_llm
{
namespace kernels
{

template <typename T>
__global__ void addBiasEndMask(T* logits, const T* bias, const int* endIds, const FinishedState* finished,
    const int* batchSlots, const int vocabSize, const int vocabSizePadded)
{
    auto const batchIdx = blockIdx.x;
    auto const batchSlot = batchSlots != nullptr ? batchSlots[batchIdx] : batchIdx;
    FinishedState const finishState = finished != nullptr ? finished[batchSlot] : FinishedState::empty();
    if (finishState.isSkipDecoding())
    {
        return;
    }

    bool finish = finishState.isFinished();
    int offset = batchIdx * vocabSizePadded;

    bool const IS_FP16 = std::is_same<T, half>::value;
    T const MAX_T_VAL = (IS_FP16) ? HALF_FLT_MAX : FLT_MAX;
    for (int tid = threadIdx.x; tid < vocabSizePadded; tid += blockDim.x)
    {
        if (tid >= vocabSize)
        {
            logits[offset + tid] = -MAX_T_VAL;
        }
        else if (finish)
        {
            logits[offset + tid] = (tid == endIds[batchSlot]) ? MAX_T_VAL : -MAX_T_VAL;
        }
        else
        {
            if (bias != nullptr)
            {
                logits[offset + tid] += bias[tid];
            }
        }
    }
}

template <typename T>
void invokeAddBiasEndMask(T* logits, const T* bias, const int* endIds, const FinishedState* finished,
    const int* batchSlots, const int batchSize, const int vocabSize, const int vocabSizePadded, hipStream_t stream)
{
    dim3 grid(batchSize);
    dim3 block(min(vocabSizePadded, 1024));
    // n is the vocabSize, e.g., 30000, 7000.... vocabSize is usually very big.
    addBiasEndMask<<<grid, block, 0, stream>>>(logits, bias, endIds, finished, batchSlots, vocabSize, vocabSizePadded);
}

template void invokeAddBiasEndMask(float* logits, const float* bias, const int* endIds, const FinishedState* finished,
    const int* batchSlots, const int batchSize, const int vocabSize, const int vocabSizePadded, hipStream_t stream);

template void invokeAddBiasEndMask(half* logits, const half* bias, const int* endIds, const FinishedState* finished,
    const int* batchSlots, const int batchSize, const int vocabSize, const int vocabSizePadded, hipStream_t stream);

template <typename T, int BLOCK_SIZE_, int BLOCKS_PER_BEAM_>
__global__ void topKStage1(const T* __restrict logProbs, T* tmpLogProbs, int* topKTmpIdBuf, T* topKTmpValBuf,
    const FinishedState* finished, const int maxTopK, const int* topKs, const int vocabSize, const int* endIds,
    const bool* skipDecode, const int* batchSlots)
{
    typedef hipcub::BlockReduce<TopK_2<T>, BLOCK_SIZE_> BlockReduce;
    __shared__ typename BlockReduce::TempStorage tempStorage;

    int const tid = threadIdx.x;
    int const bid = blockIdx.x;

    auto const batchId = bid / BLOCKS_PER_BEAM_; // row id for logProbs
    auto const batchSlot = batchSlots != nullptr ? batchSlots[batchId] : batchId;
    FinishedState const finishState = finished != nullptr ? finished[batchSlot] : FinishedState::empty();
    if ((skipDecode != nullptr && skipDecode[batchSlot]) || (finishState.isSkipDecoding()))
    {
        return;
    }
    const int blockLane = bid % BLOCKS_PER_BEAM_;                  // block id for a beam
    const int k = (topKs != nullptr) ? topKs[batchSlot] : maxTopK; // batchId = batch index

    const int logBufIndex = batchId * vocabSize;
    const int tmpLogBufIndex = batchId * vocabSize;
    const int tmpTopKBufIndex = batchId * BLOCKS_PER_BEAM_ * maxTopK + blockLane * k;

    TopK_2<T> partial;
    const bool IS_FP16 = std::is_same<T, half>::value;
    const T MAX_T_VAL = (IS_FP16) ? HALF_FLT_MAX : FLT_MAX;

    if (finished != nullptr && finishState.isFinished())
    {
        if (tid < k)
        {
            const int index = tmpTopKBufIndex + tid;
            if (blockLane == 0 && tid == 0)
            {
                const int endId = endIds[batchSlot];
                topKTmpIdBuf[index] = tmpLogBufIndex + endId;
                topKTmpValBuf[index] = logProbs[logBufIndex + endId];
            }
            else
            {
                topKTmpIdBuf[index] = -1;
                topKTmpValBuf[index] = -MAX_T_VAL;
            }
        }
        return;
    }

    for (int elemId = tid + blockLane * BLOCK_SIZE_; elemId < vocabSize; elemId += BLOCK_SIZE_ * BLOCKS_PER_BEAM_)
    {
        int localIndex = elemId + tmpLogBufIndex;
        int globalIndex = elemId + logBufIndex;
        tmpLogProbs[localIndex] = logProbs[globalIndex];
    }

    for (int ite = 0; ite < k; ite++)
    {
        partial.init();
#pragma unroll
        for (int elemId = tid + blockLane * BLOCK_SIZE_; elemId < vocabSize; elemId += BLOCK_SIZE_ * BLOCKS_PER_BEAM_)
        {
            int index = elemId + tmpLogBufIndex;
            partial.insert(tmpLogProbs[index], index);
        }

        TopK_2<T> total = BlockReduce(tempStorage).Reduce(partial, reduce_topk_op_2<T>);

        if (tid == 0)
        {
            const int index = tmpTopKBufIndex + ite;
            topKTmpIdBuf[index] = total.p;
            topKTmpValBuf[index] = total.u;
            if (total.p >= 0)
            {
                tmpLogProbs[total.p] = -MAX_T_VAL;
            }
        }
        __syncthreads();
    }
}

template <typename T, int BLOCK_SIZE_, int BLOCKS_PER_BEAM_>
__global__ void topKStage2Sampling(const int* __restrict topKTmpIdBuf, T* topKTmpValBuf, int** ids,
    int* sequenceLengths, const FinishedState* finishedInput, FinishedState* finishedOutput, float* cumLogProbs,
    float* outputLogProbs, const int maxTopK, const int* topKs, const float topP, const float* topPs,
    hiprandState_t* curandstate, const int* endIds, const int vocabSize, const bool* skipDecode, const int* batchSlots,
    const bool normalizeLogProbs)
{
    bool const IS_FP16 = std::is_same<T, half>::value;
    T const MAX_T_VAL = (IS_FP16) ? HALF_FLT_MAX : FLT_MAX;

    int const tid = threadIdx.x;
    auto const batchIdx = blockIdx.x;
    auto const batchSlot = batchSlots != nullptr ? batchSlots[batchIdx] : batchIdx;
    FinishedState const finishState = finishedInput != nullptr ? finishedInput[batchSlot] : FinishedState::empty();
    if ((skipDecode != nullptr && skipDecode[batchSlot]) || (finishState.isSkipDecoding()))
    {
        return;
    }

    const int k = (topKs != nullptr) ? topKs[batchSlot] : maxTopK;
    const float probThreshold = (topPs != nullptr) ? topPs[batchSlot] : topP;
    const int size = k * BLOCKS_PER_BEAM_;
    const int stride = maxTopK * BLOCKS_PER_BEAM_;

    typedef hipcub::BlockReduce<TopK_2<float>, BLOCK_SIZE_> BlockReduce;
    __shared__ typename BlockReduce::TempStorage tempStorage;
    extern __shared__ char array[];
    __shared__ float s_sum;
    T* s_val = topKTmpValBuf + batchIdx * stride;
    int* s_id = reinterpret_cast<int*>(array);
    if (tid == 0)
    {
        s_sum = 0.0f;
    }
    TopK_2<float> partial;

    if (finishState.isFinished())
    {
        if (finishedOutput != nullptr)
        {
            finishedOutput[batchSlot] = finishState;
        }
        return;
    }

    float* s_val2 = reinterpret_cast<float*>(s_id + k);
    float maxLogit;
    for (int ite = 0; ite < k; ite++)
    {
        partial.init();
#pragma unroll
        for (int i = tid; i < size; i += BLOCK_SIZE_)
        {
            partial.insert((float) s_val[i], i);
        }

        TopK_2<float> total = BlockReduce(tempStorage).Reduce(partial, reduce_topk_op_2<float>);

        if (tid == 0)
        {
            if (ite == 0)
            {
                maxLogit = total.u;
            }
            s_id[ite] = total.p;
            s_val[total.p] = -MAX_T_VAL;

            // when cumLogProbs are computed, topKTmpValBuf (logits_buf_) are
            // already pre-processed by softmax_kernel
            if (cumLogProbs == nullptr && outputLogProbs == nullptr)
            {
                total.u = __expf(total.u - maxLogit);
            }
            s_val2[ite] = total.u;
            s_sum += total.u;
        }
        __syncthreads();
    }

    if (tid == 0)
    {
        float randNum = (float) hiprand_uniform(curandstate + batchSlot) * probThreshold * s_sum;
        for (int i = 0; i < k; i++)
        {
            float expLogit = s_val2[i];
            randNum = randNum - expLogit;
            if (randNum <= 0.0f || i == k - 1)
            {
                int idx = s_id[i];
                // If s_id is -1 here we force output token to the last from vocabulary to get vivid indicator of smth
                // going wrong for the debug
                auto outputId = idx != -1 ? topKTmpIdBuf[batchIdx * stride + idx] % vocabSize : vocabSize - 1;
                ids[batchSlot][sequenceLengths[batchSlot]] = outputId;
                if (cumLogProbs != nullptr || outputLogProbs != nullptr)
                {
                    float logProb = logf(expLogit);
                    if (cumLogProbs != nullptr)
                    {
                        cumLogProbs[batchSlot] += logProb;
                    }
                    if (outputLogProbs != nullptr)
                    {
                        // 'outputLogProbs' is the probability induced by the top-k sampling:
                        // NOT normalized (same way as OpenAI does):
                        // log_prob = log P(i | i is in top-k) = log(expLogit)
                        // normalized:
                        // log_prob = log P(i | i is in top-k) = log(expLogit / sum)
                        outputLogProbs[batchSlot] = normalizeLogProbs ? logProb - logf(s_sum) : logProb;
                    }
                }
                break;
            }
        }
        if (sequenceLengths != nullptr && finishedOutput != nullptr)
        {
            const int seqLen = sequenceLengths[batchSlot];
            if (ids[batchSlot][seqLen] == endIds[batchSlot])
            {
                finishedOutput[batchSlot].setFinishedEOS();
                // Do not increase seq len when EOS is generated. Seq len should always contain only tokens to be
                // outputted
            }
            else
            {
                // We don't need to set output finished state as it is assumed to be in non finished state
                sequenceLengths[batchSlot] += 1;
            }
        }
    }
}

#define CASE_K(K_MAX, BLOCK_SIZE_1_, BLOCK_SIZE_2_, BLOCKS_PER_BEAM_, normalizeLogProbs)                               \
    topKStage1<T, BLOCK_SIZE_1_, BLOCKS_PER_BEAM_>                                                                     \
        <<<batchSize * BLOCKS_PER_BEAM_, BLOCK_SIZE_1_, 0, stream>>>(logProbs, tempLogProbs, topKTmpIdBuf,             \
            topKTmpValBuf, finishedInput, maxTopK, topKs, vocabSize, endIds, skipDecode, batchSlots);                  \
    topKStage2Sampling<T, BLOCK_SIZE_2_, BLOCKS_PER_BEAM_>                                                             \
        <<<batchSize, BLOCK_SIZE_2_, K_MAX * sizeof(int) + K_MAX * sizeof(float), stream>>>(topKTmpIdBuf,              \
            topKTmpValBuf, ids, sequenceLengths, finishedInput, finishedOutput, cumLogProbs, outputLogProbs, maxTopK,  \
            topKs, topP, topPs, curandstate, endIds, vocabSize, skipDecode, batchSlots, normalizeLogProbs);            \
    break;

template <typename T>
void invokeBatchTopKSampling(void* workspace, size_t& workspaceSize, const T* logProbs, int** ids, int* sequenceLengths,
    const FinishedState* finishedInput, FinishedState* finishedOutput, float* cumLogProbs, float* outputLogProbs,
    hiprandState_t* curandstate, const int maxTopK, const int* topKs, const float topP, const float* topPs,
    const int vocabSizePadded, const int* endIds, const int* batchSlots, hipStream_t stream, const int batchSize,
    const bool* skipDecode, const bool normalizeLogProbs)
{
    TLLM_LOG_DEBUG("%s start", __PRETTY_FUNCTION__);

    // Not allow an ambiguous inputs topP and topPs.
    assert(topP == 1.0f || topPs == nullptr);
    const int vocabSize = vocabSizePadded;
    const int maxBlockPerBeam = 8;
    int tempLogProbsBufSize = batchSize * vocabSize;                // type float
    int topKTmpIdsBufSize = batchSize * maxTopK * maxBlockPerBeam;  // type int
    int topKTmpValBuf_size = batchSize * maxTopK * maxBlockPerBeam; // type float

    // prevent memory misaligned address
    tempLogProbsBufSize = (int) (ceil(tempLogProbsBufSize / 4.)) * 4;
    topKTmpIdsBufSize = (int) (ceil(topKTmpIdsBufSize / 4.)) * 4;
    topKTmpValBuf_size = (int) (ceil(topKTmpValBuf_size / 4.)) * 4;

    if (workspace == nullptr)
    {
        workspaceSize
            = sizeof(T) * tempLogProbsBufSize + sizeof(int) * topKTmpIdsBufSize + sizeof(T) * topKTmpValBuf_size;
        return;
    }

    T* tempLogProbs = (T*) workspace;
    int* topKTmpIdBuf = (int*) (tempLogProbs + tempLogProbsBufSize);
    T* topKTmpValBuf = (T*) (topKTmpIdBuf + topKTmpIdsBufSize);

    int logMaxTopK(0);
    int recursor(maxTopK - 1);
    while (recursor >>= 1)
        ++logMaxTopK;
    switch (logMaxTopK)
    {
    case 0:
    case 1:
    case 2:
    case 3: // 0 < maxTopK <= 16
        CASE_K(16, 128, 128, 8, normalizeLogProbs);
    case 4: // 16 < maxTopK <= 32
        CASE_K(32, 256, 128, 8, normalizeLogProbs);
    case 5: // 32 < maxTopK <= 64
        CASE_K(64, 256, 256, 8, normalizeLogProbs);
    case 6:
    case 7:
    case 8:
    case 9: // 64 < maxTopK <= 1024
        CASE_K(1024, 256, 256, 8, normalizeLogProbs);
    default: throw std::domain_error(fmtstr("top-k kernel supports 1<=k<=1024 but got k=%d", maxTopK));
    }
}

#undef CASE_K

template void invokeBatchTopKSampling(void* workspace, size_t& workspaceSize, const float* logProbs, int** ids,
    int* sequenceLengths, const FinishedState* finishedInput, FinishedState* finishedOutput, float* cumLogProbs,
    float* outputLogProbs, hiprandState_t* curandstate, const int maxTopK, const int* topKs, const float topP,
    const float* topPs, const int vocabSizePadded, const int* endIds, const int* batchSlots, hipStream_t stream,
    const int batchSize, const bool* skipDecode, const bool normalizeLogProbs);

template void invokeBatchTopKSampling(void* workspace, size_t& workspaceSize, const half* logProbs, int** ids,
    int* sequenceLengths, const FinishedState* finishedInput, FinishedState* finishedOutput, float* cumLogProbs,
    float* outputLogProbs, hiprandState_t* curandstate, const int maxTopK, const int* topKs, const float topP,
    const float* topPs, const int vocabSizePadded, const int* endIds, const int* batchSlots, hipStream_t stream,
    const int batchSize, const bool* skipDecode, const bool normalizeLogProbs);

template <typename T>
void invokeTopKSampling(void* workspace, size_t& workspaceSize, const T* logProbs, int** ids, int* sequenceLengths,
    const FinishedState* finishedInput, FinishedState* finishedOutput, float* cumLogProbs, float* outputLogProbs,
    hiprandState_t* curandstate, const int topK, const float topP, const int vocabSizePadded, const int* endIds,
    const int* batchSlots, hipStream_t stream, const int batchSize, const bool* skipDecode,
    const bool normalizeLogProbs)
{
    invokeBatchTopKSampling(workspace, workspaceSize, logProbs, ids, sequenceLengths, finishedInput, finishedOutput,
        cumLogProbs, outputLogProbs, curandstate, topK, nullptr, topP, nullptr, vocabSizePadded, endIds, batchSlots,
        stream, batchSize, skipDecode, normalizeLogProbs);
}

template void invokeTopKSampling(void* workspace, size_t& workspaceSize, const float* logProbs, int** ids,
    int* sequenceLengths, const FinishedState* finishedInput, FinishedState* finishedOutput, float* cumLogProbs,
    float* outputLogProbs, hiprandState_t* curandstate, const int topK, const float topP, const int vocabSizePadded,
    const int* endIds, const int* batchSlots, hipStream_t stream, const int batchSize, const bool* skipDecode,
    const bool normalizeLogProbs);

template void invokeTopKSampling(void* workspace, size_t& workspaceSize, const half* logProbs, int** ids,
    int* sequenceLengths, const FinishedState* finishedInput, FinishedState* finishedOutput, float* cumLogProbs,
    float* outputLogProbs, hiprandState_t* curandstate, const int topK, const float topP, const int vocabSizePadded,
    const int* endIds, const int* batchSlots, hipStream_t stream, const int batchSize, const bool* skipDecode,
    const bool normalizeLogProbs);

} // namespace kernels
} // namespace tensorrt_llm
