#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2023, NVIDIA CORPORATION.  All rights reserved.
 * Copyright (c) 2021, NAVER Corp.  Authored by CLOVA.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "tensorrt_llm/common/logger.h"
#include "tensorrt_llm/common/memoryUtils.h"
#include "tensorrt_llm/kernels/decodingCommon.h"
#include "tensorrt_llm/kernels/samplingTopKKernels.h"
#include "tensorrt_llm/kernels/samplingTopPKernels.h"
#include "tensorrt_llm/layers/topKSamplingLayer.h"
#include "tensorrt_llm/runtime/iTensor.h"

#include <algorithm>
#include <float.h>

using namespace tensorrt_llm::common;
using namespace tensorrt_llm::kernels;
using namespace tensorrt_llm::runtime;

namespace tensorrt_llm
{
namespace layers
{

template <uint32_t TOP_K_MAX>
__global__ void setupTopKRuntimeArgs(int batchSize, uint32_t topK, uint32_t* topKs, int topKsSize, float topP,
    float* topPs, int topPsSize, bool* skipDecode, const int* batchSlots)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    for (int bi = index; bi < batchSize; bi += gridDim.x * blockDim.x)
    {
        auto const batchSlot = batchSlots != nullptr ? batchSlots[bi] : bi;
        uint32_t k = topKsSize > 1 ? topKs[batchSlot] : topK;
        float p = topPsSize > 1 ? topPs[batchSlot] : topP;
        if (k == 0 && p == 0.0f)
        {
            // TensorRT-LLM's topp implementation does not support topp = 0.0f, but it
            // equivalent to greedy search. So, we set the topk = 1 as an alternative
            // solution.
            k = 1;
        }
        if (k > 0 && p == 0.0f)
        {
            // This case corresponds to the old topk sampling, which is equivalent to
            // the old topk_topp sampling with topp=1.0f. TopKSamplingLayer and
            // TopKTopPSamplingLayer are now merged by TopKSamplingLayer. Thus, we
            // replace the case topk>0 and topp=0.0f by topk>0 and topp=1.0f for the
            // compatibility.
            p = 1.0f;
        }
        // Clip k value. A topk sampling kernel supports up to TOP_K_MAX.
        topKs[batchSlot] = k;
        // Clip p value if it is out of range. range = [0.0, 1.0].
        topPs[batchSlot] = p;
        skipDecode[batchSlot] = k == 0;
    }
}

template <typename T>
void TopKSamplingLayer<T>::allocateBuffer(size_t const batchSize)
{
    TLLM_LOG_TRACE(__PRETTY_FUNCTION__);
    invokeTopKSampling<T>(nullptr, mSamplingWorkspaceSize, nullptr, nullptr, nullptr, nullptr, nullptr, nullptr,
        nullptr, nullptr, TOP_K_MAX, 1.0f, mVocabSizePadded, nullptr, nullptr, mStream, batchSize, mSkipDecodeDevice,
        mNormalizeLogProbs);

    std::array<size_t, 4> deviceBufferSizes;
    deviceBufferSizes[0] = mSamplingWorkspaceSize;
    deviceBufferSizes[1] = sizeof(uint32_t) * batchSize;
    deviceBufferSizes[2] = sizeof(float) * batchSize;
    deviceBufferSizes[3] = std::max(deviceBufferSizes[1], deviceBufferSizes[2]);

    mSamplingWorkspaceDevice = mAllocator->reMalloc(mSamplingWorkspaceDevice, deviceBufferSizes[0], false);
    mRuntimeTopKDevice = mAllocator->reMalloc(mRuntimeTopKDevice, deviceBufferSizes[1], false);
    mRuntimeTopPDevice = mAllocator->reMalloc(mRuntimeTopPDevice, deviceBufferSizes[2], false);
    mSetupWorkspaceDevice = mAllocator->reMalloc(mSetupWorkspaceDevice, deviceBufferSizes[3], false);

    auto const bytesAllocated = std::accumulate(deviceBufferSizes.begin(), deviceBufferSizes.end(), 0);
    TLLM_LOG_DEBUG("topKSamplingLayer allocated %d bytes on GPU", bytesAllocated);

    mIsAllocateBuffer = true;
}

template <typename T>
void TopKSamplingLayer<T>::freeBuffer()
{
    TLLM_LOG_TRACE(__PRETTY_FUNCTION__);
    if (mIsAllocateBuffer)
    {
        mAllocator->free((void**) (&mSamplingWorkspaceDevice));
        mAllocator->free((void**) (&mRuntimeTopKDevice));
        mAllocator->free((void**) (&mRuntimeTopPDevice));
        mAllocator->free((void**) (&mSetupWorkspaceDevice));
    }
    BaseSamplingLayer<T>::freeBuffer();
    mIsAllocateBuffer = false;
}

template <typename T>
void TopKSamplingLayer<T>::setup(size_t const batchSize, int const* batchSlots, SetupParams const& setupParams)
{
    TLLM_LOG_TRACE(__PRETTY_FUNCTION__);
    BaseSamplingLayer<T>::setupBase(batchSize, batchSlots, setupParams);

    uint32_t constexpr defaultTopK = 0;
    auto runtimeTopK = setupParams.runtime_top_k.value_or(std::vector<uint32_t>{defaultTopK});
    auto runtimeTopP = setupParams.runtime_top_p.value_or(std::vector<float>{});

    size_t const runtimeTopKSize = runtimeTopK.size();
    size_t const runtimeTopPSize = runtimeTopP.size();
    mNormalizeLogProbs = setupParams.normalize_log_probs.has_value() && setupParams.normalize_log_probs.value();

    for (auto& topP : runtimeTopP)
    {
        if (topP < 0.f || topP > 1.0f)
        {
            TLLM_LOG_WARNING("TopP (%f) is out of range ([0.0, 1.0f]). Clip to closest number.", topP);
            topP = std::clamp(topP, 0.f, 1.f);
        }
    }
    for (auto& topK : runtimeTopK)
    {
        if (topK > TOP_K_MAX)
        {
            TLLM_LOG_WARNING(
                "TopK (%d) is larger than max supported number (%d). Clip to max supported number.", topK, TOP_K_MAX);
            topK = TOP_K_MAX;
        }
    }

    uint32_t const topK = *std::max_element(std::begin(runtimeTopK), std::end(runtimeTopK));
    float const topP = (runtimeTopPSize == 0) ? 0.0f : runtimeTopP.front();

    if (runtimeTopKSize > 1)
    {
        TLLM_CHECK_WITH_INFO(runtimeTopK.size() == batchSize,
            fmtstr("runtimeTopK.size() (%lu) == batchSize (%lu) is not satisfied!", runtimeTopK.size(), batchSize));
        cudaAutoCpy(reinterpret_cast<uint32_t*>(mSetupWorkspaceDevice), runtimeTopK.data(), batchSize, mStream);
        invokeScatterDecodingParams(
            reinterpret_cast<uint32_t*>(mSetupWorkspaceDevice), mRuntimeTopKDevice, batchSlots, batchSize, mStream);
    }
    if (runtimeTopPSize > 1)
    {
        TLLM_CHECK_WITH_INFO(runtimeTopP.size() == batchSize,
            fmtstr("runtimeTopP.size() (%lu) == batchSize (%lu) is not satisfied!", runtimeTopP.size(), batchSize));
        cudaAutoCpy(reinterpret_cast<float*>(mSetupWorkspaceDevice), runtimeTopP.data(), batchSize, mStream);
        invokeScatterDecodingParams(
            reinterpret_cast<float*>(mSetupWorkspaceDevice), mRuntimeTopPDevice, batchSlots, batchSize, mStream);
    }

    dim3 block(std::min((int) batchSize, 256));
    dim3 grid(divUp((int) batchSize, (int) block.x));
    // support topK up to TOP_K_MAX.
    setupTopKRuntimeArgs<TOP_K_MAX><<<grid, block, 0, mStream>>>(batchSize, topK, mRuntimeTopKDevice, runtimeTopKSize,
        topP, mRuntimeTopPDevice, runtimeTopPSize, mSkipDecodeDevice, batchSlots);
    cudaAutoCpy(mSkipDecodeHost, mSkipDecodeDevice, mMaxBatchSize, mStream);
    std::vector<uint32_t> runtimeTopKs(mMaxBatchSize);
    cudaAutoCpy(runtimeTopKs.data(), mRuntimeTopKDevice, mMaxBatchSize, mStream);
    // TODO(nkorobov): find maxTopK using batch slot
    mRuntimeMaxTopK = *std::max_element(std::begin(runtimeTopKs), std::end(runtimeTopKs));
}

template <typename T>
void TopKSamplingLayer<T>::runSampling(DecodingOutputParams& outputs, DecodingParams const& inputs)
{
    TLLM_LOG_TRACE("%s start", __PRETTY_FUNCTION__);

    auto const batchSize = inputs.logits.shape[0];

    // in case of skip any, the logit value is already copied and processed.
    auto* logits = mSkipAny ? mRuntimeLogitsDevice : inputs.logits.template getPtr<T>();
    auto* endIds = inputs.end_ids.template getPtr<const int>();
    auto* batchSlots = inputs.batch_slots ? inputs.batch_slots->template getPtr<const int>() : nullptr;

    FinishedState* finishedInput = (inputs.finished)
        ? reinterpret_cast<FinishedState*>(inputs.finished->template getPtr<FinishedState::UnderlyingType>())
        : nullptr;
    FinishedState* finishedOutput = (outputs.finished)
        ? reinterpret_cast<FinishedState*>(outputs.finished->template getPtr<FinishedState::UnderlyingType>())
        : nullptr;
    invokeAddBiasEndMask(
        logits, (T*) (nullptr), endIds, finishedInput, batchSlots, batchSize, mVocabSize, mVocabSizePadded, mStream);
    sync_check_cuda_error();

    float* cumLogProbs = (outputs.cum_log_probs) ? outputs.cum_log_probs->template getPtr<float>() : nullptr;
    float* outputLogProbs = (outputs.output_log_probs) ? outputs.output_log_probs->template getPtr<float>() : nullptr;

    if (cumLogProbs != nullptr || outputLogProbs != nullptr)
    {
        invokeAddBiasSoftMax(logits, logits, (T*) (nullptr), endIds, finishedInput, batchSlots, batchSize, mVocabSize,
            mVocabSizePadded, mStream);
        sync_check_cuda_error();
    }

    int* sequenceLength = (outputs.sequence_length) ? outputs.sequence_length->template getPtr<int>() : nullptr;

    invokeBatchTopKSampling(mSamplingWorkspaceDevice, mSamplingWorkspaceSize, logits,
        outputs.output_ids_ptr.template getPtr<int*>(), sequenceLength, finishedInput, finishedOutput, cumLogProbs,
        outputLogProbs, mCurandStatesDevice,
        (int) mRuntimeMaxTopK, // useless because mRuntimeTopKDevice is never
                               // nullptr. Keep for legacy.
        (int*) (mRuntimeTopKDevice),
        1.0f,                  // useless because mRuntimeTopPDevice is never nullptr. Keep for
                               // legacy.
        mRuntimeTopPDevice, mVocabSizePadded, endIds, batchSlots, mStream, batchSize, mSkipDecodeDevice,
        mNormalizeLogProbs);
    sync_check_cuda_error();
}

template <typename T>
TopKSamplingLayer<T>::TopKSamplingLayer(size_t maxBatchSize, size_t vocabSize, size_t vocabSizePadded,
    hipStream_t stream, std::shared_ptr<IAllocator> allocator)
    : BaseSamplingLayer<T>(maxBatchSize, vocabSize, vocabSizePadded, stream, std::move(allocator), nullptr)
{
    allocateBuffer(mMaxBatchSize);
}

template <typename T>
TopKSamplingLayer<T>::TopKSamplingLayer(TopKSamplingLayer<T> const& topKSamplingLayer)
    : BaseSamplingLayer<T>(topKSamplingLayer)
{
    allocateBuffer(mMaxBatchSize);
}

template <typename T>
TopKSamplingLayer<T>::~TopKSamplingLayer()
{
    TLLM_LOG_TRACE(__PRETTY_FUNCTION__);
    freeBuffer();
}

template class TopKSamplingLayer<float>;
template class TopKSamplingLayer<half>;

} // namespace layers
} // namespace tensorrt_llm
