#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2023, NVIDIA CORPORATION.  All rights reserved.
 * Copyright (c) 2021, NAVER Corp.  Authored by CLOVA.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "tensorrt_llm/common/logger.h"
#include "tensorrt_llm/common/memoryUtils.h"
#include "tensorrt_llm/common/reduceKernelUtils.cuh"
#include "tensorrt_llm/kernels/decodingCommon.h"
#include "tensorrt_llm/kernels/samplingTopKKernels.h"
#include "tensorrt_llm/kernels/samplingTopPKernels.h"
#include "tensorrt_llm/layers/topPSamplingLayer.h"

#include <algorithm>
#include <float.h>

using namespace tensorrt_llm::common;
using namespace tensorrt_llm::kernels;

namespace tensorrt_llm
{
namespace layers
{

static __global__ void set_topp_runtime_args(int batchSize, uint32_t top_k, uint32_t* top_ks, int top_ks_size,
    float top_p, float* top_ps, int top_ps_size, bool* skip_decode, float* initial_top_p_buf, float* top_p_decay_buf,
    float* top_p_min_buf, const int* batch_slots)
{
    /**
     * @brief Setup the runtime arguments for topp, broadcasting top_p to top_ps
              and top_k to top_ks, verifying value ranges of top_p_decay/top_p_min.
     *
     * \param batchSize
     * \param top_k
     * \param top_ks                [batchSize]
     * \param top_ks_size
     * \param top_p
     * \param top_ps                [batchSize]
     * \param top_ps_size
     * \param skip_decode           [batchSize]
     * \param initial_top_p_buf     [batchSize]
     * \param top_p_decay_buf       [batchSize]
     * \param top_p_min_buf         [batchSize]
     *
     */

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    for (int bi = index; bi < batchSize; bi += gridDim.x * blockDim.x)
    {
        auto const batch_slot = batch_slots != nullptr ? batch_slots[bi] : bi;
        std::uint32_t k = top_ks_size > 1 ? top_ks[batch_slot] : top_k;
        float p = top_ps_size > 1 ? top_ps[batch_slot] : top_p;
        if (k == 0 && p == 0.0f)
        {
            // TensorRT-LLM's topp implementation does not support topp = 0.0f, but it
            // equivalent to greedy search. So, we set the topk = 1 as an alternative
            // solution.
            k = 1;
        }
        top_ks[batch_slot] = k;
        top_ps[batch_slot] = p;
        skip_decode[batch_slot] = k > 0;

        initial_top_p_buf[batch_slot] = top_ps[batch_slot];
    }
}

template <typename T>
void TopPSamplingLayer<T>::allocateBuffer(size_t batchSize)
{
    TLLM_LOG_TRACE(__PRETTY_FUNCTION__);
    if (is_deterministic_)
    {
        invokeTopPSampling<T>(nullptr, // workspace
            mSamplingWorkspaceSize, cub_temp_storage_size_,
            nullptr,                   // output_ids
            nullptr,                   // sequence_length
            nullptr,                   // finished_input_buffer
            nullptr,                   // finished_output_buffer
            nullptr,                   // cum_log_probs
            nullptr,                   // output_log_probs
            nullptr,                   // log_probs
            topp_id_vals_buf_, topp_offset_buf_, begin_topp_offset_buf_, mCurandStatesDevice, batchSize,
            mVocabSizePadded, nullptr, 0.f, mStream, mSkipDecodeDevice, nullptr);
    }
    else
    {
        invokeAirTopPSampling<T>(nullptr, mSamplingWorkspaceSize,
            nullptr, // output_ids
            nullptr, // sequence_length
            nullptr, // finished_input_buffer
            nullptr, // finished_output_buffer
            nullptr, // cum_log_probs
            nullptr, // output_log_probs
            nullptr, // log_probs)
            mCurandStatesDevice, batchSize, mVocabSizePadded, nullptr, 0.f, mStream, air_topp_block_num_,
            mSkipDecodeDevice, nullptr);
    }

    std::array<size_t, 11> deviceBufferSizes;
    deviceBufferSizes[0] = mSamplingWorkspaceSize;
    deviceBufferSizes[1] = sizeof(int32_t) * batchSize * mVocabSizePadded;
    deviceBufferSizes[2] = sizeof(int32_t) * (batchSize + 1);
    deviceBufferSizes[3] = sizeof(int32_t) * (batchSize + 1);
    deviceBufferSizes[4] = sizeof(uint32_t) * batchSize;
    deviceBufferSizes[5] = sizeof(float) * batchSize;
    deviceBufferSizes[6] = sizeof(float) * batchSize;
    deviceBufferSizes[7] = sizeof(float) * batchSize;
    deviceBufferSizes[8] = sizeof(float) * batchSize;
    deviceBufferSizes[9] = sizeof(int32_t) * batchSize;
    deviceBufferSizes[10] = *std::max_element(&deviceBufferSizes[4], &deviceBufferSizes[10]);

    mSamplingWorkspaceDevice = mAllocator->reMalloc(mSamplingWorkspaceDevice, deviceBufferSizes[0], true);
    topp_id_vals_buf_ = mAllocator->reMalloc(topp_id_vals_buf_, deviceBufferSizes[1], false);
    topp_offset_buf_ = mAllocator->reMalloc(topp_offset_buf_, deviceBufferSizes[2], false);
    begin_topp_offset_buf_ = mAllocator->reMalloc(begin_topp_offset_buf_, deviceBufferSizes[3], false);
    runtime_top_k_buf_ = mAllocator->reMalloc(runtime_top_k_buf_, deviceBufferSizes[4], false);
    runtime_top_p_buf_ = mAllocator->reMalloc(runtime_top_p_buf_, deviceBufferSizes[5], false);
    initial_top_p_buf_ = mAllocator->reMalloc(initial_top_p_buf_, deviceBufferSizes[6], false);
    top_p_decay_buf_ = mAllocator->reMalloc(top_p_decay_buf_, deviceBufferSizes[7], false);
    top_p_min_buf_ = mAllocator->reMalloc(top_p_min_buf_, deviceBufferSizes[8], false);
    top_p_reset_ids_buf_ = mAllocator->reMalloc(top_p_reset_ids_buf_, deviceBufferSizes[9], false);
    setup_workspace_buf_ = mAllocator->reMalloc(setup_workspace_buf_, deviceBufferSizes[10], false);

    auto const bytesAllocated = std::accumulate(deviceBufferSizes.begin(), deviceBufferSizes.end(), 0);
    TLLM_LOG_DEBUG("topPSamplingLayer allocated %d bytes on GPU", bytesAllocated);

    mIsAllocateBuffer = true;
}

template <typename T>
void TopPSamplingLayer<T>::freeBuffer()
{
    TLLM_LOG_TRACE(__PRETTY_FUNCTION__);
    if (mIsAllocateBuffer)
    {
        mAllocator->free((void**) (&mSamplingWorkspaceDevice));
        mAllocator->free((void**) (&topp_id_vals_buf_));
        mAllocator->free((void**) (&topp_offset_buf_));
        mAllocator->free((void**) (&begin_topp_offset_buf_));
        mAllocator->free((void**) (&runtime_top_k_buf_));
        mAllocator->free((void**) (&runtime_top_p_buf_));
        mAllocator->free((void**) (&initial_top_p_buf_));
        mAllocator->free((void**) (&top_p_decay_buf_));
        mAllocator->free((void**) (&top_p_min_buf_));
        mAllocator->free((void**) (&top_p_reset_ids_buf_));
        mAllocator->free((void**) (&setup_workspace_buf_));
    }
    BaseSamplingLayer<T>::freeBuffer();
    mIsAllocateBuffer = false;
}

template <typename T>
void TopPSamplingLayer<T>::setup(size_t const batchSize, int const* batchSlots, SetupParams const& setupParams)
{
    TLLM_LOG_TRACE(__PRETTY_FUNCTION__);
    BaseSamplingLayer<T>::setupBase(batchSize, batchSlots, setupParams);

    uint32_t const defaultTopK = 0;
    auto runtimeTopK = setupParams.runtime_top_k.value_or(std::vector<uint32_t>{defaultTopK});
    auto runtimeTopP = setupParams.runtime_top_p.value_or(std::vector<float>{});

    size_t const runtimeTopKSize = runtimeTopK.size();
    size_t const runtimeTopPSize = runtimeTopP.size();

    float const defaultTopPDecay{1.0f};
    auto decayVec = setupParams.top_p_decay.value_or(std::vector<float>(batchSize, defaultTopPDecay));

    float const defaultTopPMin{1e-6f}; // prevent topp becoming 0.0
    auto topPMinVec = setupParams.top_p_min.value_or(std::vector<float>(batchSize, defaultTopPMin));

    int32_t const defaultTopPResetId{-1};
    auto topPResetIdsVec = setupParams.top_p_reset_ids.value_or(std::vector<int32_t>(batchSize, defaultTopPResetId));

    if (runtimeTopPSize == 0)
    {
        std::fill_n(mSkipDecodeHost, batchSize, true);
        return;
    }

    for (auto& topP : runtimeTopP)
    {
        if (topP < 0.f || topP > 1.0f)
        {
            TLLM_LOG_WARNING("TopP (%f) is out of range ([0.0, 1.0f]). Clip to closest number.", topP);
            topP = std::clamp(topP, 0.f, 1.f);
        }
    }

    for (auto& decay : decayVec)
    {
        if (decay <= 0.f || decay > 1.0f)
        {
            TLLM_LOG_WARNING("Decay (%f) is out of range ([0.0, 1.0f]). Change to 1.0.", decay);
            decay = 1.0f;
        }
    }

    for (auto& topPMin : topPMinVec)
    {
        if (topPMin <= 0.f || topPMin > 1.0f)
        {
            TLLM_LOG_WARNING("TopP min (%f) is out of range ([0.0, 1.0f]). Change to 0.5.", topPMin);
            topPMin = 0.5f;
        }
    }

    uint32_t const topK = runtimeTopK.at(0);
    float const topP = runtimeTopP.at(0);

    if (runtimeTopKSize > 1)
    {
        TLLM_CHECK_WITH_INFO(runtimeTopK.size() == batchSize,
            fmtstr("runtimeTopK.size() (%lu) == batchSize (%lu) is not satisfied!", runtimeTopK.size(), batchSize));
        cudaAutoCpy(reinterpret_cast<uint32_t*>(setup_workspace_buf_), runtimeTopK.data(), batchSize, mStream);
        invokeScatterDecodingParams(
            reinterpret_cast<uint32_t*>(setup_workspace_buf_), runtime_top_k_buf_, batchSlots, batchSize, mStream);
    }
    if (runtimeTopPSize > 1)
    {
        TLLM_CHECK_WITH_INFO(runtimeTopP.size() == batchSize,
            fmtstr("runtime_top_p.size() (%lu) == batchSize (%lu) is not satisfied!", runtimeTopP.size(), batchSize));
        cudaAutoCpy(reinterpret_cast<float*>(setup_workspace_buf_), runtimeTopP.data(), batchSize, mStream);
        invokeScatterDecodingParams(
            reinterpret_cast<float*>(setup_workspace_buf_), runtime_top_p_buf_, batchSlots, batchSize, mStream);
    }

    auto fillBuffers
        = [this, &batchSize, &batchSlots](std::string name, auto const& vector, auto deviceTmpBuffer, auto deviceBuffer)
    {
        TLLM_CHECK_WITH_INFO(vector.size() == batchSize,
            fmtstr("%s.size() (%lu) == batchSize (%lu) is not satisfied!", name.c_str(), vector.size(), batchSize));
        cudaAutoCpy(deviceTmpBuffer, vector.data(), batchSize, mStream);
        invokeScatterDecodingParams(deviceTmpBuffer, deviceBuffer, batchSlots, batchSize, mStream);
    };

    fillBuffers("top_p_decay", decayVec, reinterpret_cast<float*>(setup_workspace_buf_), top_p_decay_buf_);

    fillBuffers("top_p_min", topPMinVec, reinterpret_cast<float*>(setup_workspace_buf_), top_p_min_buf_);

    fillBuffers(
        "top_p_reset_ids", topPResetIdsVec, reinterpret_cast<int32_t*>(setup_workspace_buf_), top_p_reset_ids_buf_);

    dim3 block(std::min((int) batchSize, 256));
    dim3 grid(divUp((int) batchSize, (int) block.x));
    set_topp_runtime_args<<<grid, block, 0, mStream>>>(batchSize, topK, runtime_top_k_buf_, runtimeTopKSize, topP,
        runtime_top_p_buf_, runtimeTopPSize, mSkipDecodeDevice, initial_top_p_buf_, top_p_decay_buf_, top_p_min_buf_,
        batchSlots);
    sync_check_cuda_error();

    cudaAutoCpy(mSkipDecodeHost, mSkipDecodeDevice, mMaxBatchSize, mStream);

    std::vector<float> runtime_top_ps(mMaxBatchSize);
    cudaAutoCpy(runtime_top_ps.data(), runtime_top_p_buf_, mMaxBatchSize, mStream);
    // TODO(nkorobov): find maxTopP using batch slots
    mRuntimeMaxTopP = *std::max_element(std::begin(runtime_top_ps), std::end(runtime_top_ps));

    if (!is_deterministic_)
    {
        int smCnt = mCudaDeviceProp->multiProcessorCount;
        air_topp_block_num_ = calcAirTopPBlockNum<T, int, float>(batchSize, (int) mVocabSizePadded, smCnt);
    }
}

template <typename T>
void TopPSamplingLayer<T>::runSampling(DecodingOutputParams& outputs, DecodingParams const& inputs)
{
    TLLM_LOG_TRACE(__PRETTY_FUNCTION__);

    auto const batchSize = inputs.logits.shape[0];

    // in case of skip any, the logit value is already copied and processed.
    auto* logits = !mSkipAny ? inputs.logits.template getPtr<T>() : mRuntimeLogitsDevice;
    auto* endIds = inputs.end_ids.template getPtr<const int>();
    auto* batchSlots = inputs.batch_slots ? inputs.batch_slots->template getPtr<const int>() : nullptr;

    if (is_deterministic_)
    {
        invokeTopPInitialize(
            topp_id_vals_buf_, topp_offset_buf_, begin_topp_offset_buf_, batchSize, mVocabSizePadded, mStream);
        sync_check_cuda_error();
    }

    FinishedState* finishedInput = (inputs.finished)
        ? reinterpret_cast<FinishedState*>(inputs.finished->template getPtr<FinishedState::UnderlyingType>())
        : nullptr;
    FinishedState* finishedOutput = (outputs.finished)
        ? reinterpret_cast<FinishedState*>(outputs.finished->template getPtr<FinishedState::UnderlyingType>())
        : nullptr;
    invokeAddBiasSoftMax(logits, logits, (T*) (nullptr), endIds, finishedInput, batchSlots, batchSize, mVocabSize,
        mVocabSizePadded, mStream);
    sync_check_cuda_error();

    float* cumLogProbs = (outputs.cum_log_probs) ? outputs.cum_log_probs->template getPtr<float>() : nullptr;
    float* outputLogProbs = (outputs.output_log_probs) ? outputs.output_log_probs->template getPtr<float>() : nullptr;
    int* sequenceLength = (outputs.sequence_length) ? outputs.sequence_length->template getPtr<int>() : nullptr;

    if (is_deterministic_)
    {
        invokeBatchTopPSampling<T>(mSamplingWorkspaceDevice, mSamplingWorkspaceSize, cub_temp_storage_size_,
            outputs.output_ids_ptr.template getPtr<int*>(), sequenceLength, finishedInput, finishedOutput, cumLogProbs,
            outputLogProbs, logits, topp_id_vals_buf_, topp_offset_buf_, begin_topp_offset_buf_, mCurandStatesDevice,
            batchSize, mVocabSizePadded, endIds, mRuntimeMaxTopP, runtime_top_p_buf_, mStream, mSkipDecodeDevice,
            batchSlots);
        sync_check_cuda_error();
        invokeComputeToppDecay(runtime_top_p_buf_, initial_top_p_buf_,
            outputs.output_ids_ptr.template getPtr<const int*>(), top_p_decay_buf_, top_p_min_buf_,
            top_p_reset_ids_buf_, sequenceLength, batchSlots, batchSize, mStream);
        sync_check_cuda_error();
    }
    else
    {
        invokeBatchAirTopPSampling<T>(mSamplingWorkspaceDevice, mSamplingWorkspaceSize,
            outputs.output_ids_ptr.template getPtr<int*>(), sequenceLength, finishedInput, finishedOutput, cumLogProbs,
            outputLogProbs, logits, mCurandStatesDevice, batchSize, mVocabSizePadded, endIds, mRuntimeMaxTopP,
            runtime_top_p_buf_, mStream, air_topp_block_num_, mSkipDecodeDevice, batchSlots);
        sync_check_cuda_error();
    }
}

template <typename T>
TopPSamplingLayer<T>::TopPSamplingLayer(std::size_t maxBatchSize, std::size_t vocabSize, std::size_t vocabSizePadded,
    hipStream_t stream, std::shared_ptr<IAllocator> allocator, hipDeviceProp_t* prop, bool isDeterministic)
    : BaseSamplingLayer<T>(maxBatchSize, vocabSize, vocabSizePadded, stream, std::move(allocator), prop)
    , is_deterministic_(isDeterministic)
{
    allocateBuffer(mMaxBatchSize);
}

template <typename T>
TopPSamplingLayer<T>::TopPSamplingLayer(TopPSamplingLayer<T> const& top_p_sampling_layer)
    : BaseSamplingLayer<T>(top_p_sampling_layer)
{
    allocateBuffer(mMaxBatchSize);
}

template <typename T>
TopPSamplingLayer<T>::~TopPSamplingLayer()
{
    TLLM_LOG_TRACE(__PRETTY_FUNCTION__);
    freeBuffer();
}

template class TopPSamplingLayer<float>;
template class TopPSamplingLayer<half>;

} // namespace layers
} // namespace tensorrt_llm
