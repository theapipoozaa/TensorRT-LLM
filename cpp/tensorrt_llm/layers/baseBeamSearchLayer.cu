#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "tensorrt_llm/common/cudaUtils.h"
#include "tensorrt_llm/common/memoryUtils.h"
#include "tensorrt_llm/kernels/penaltyKernels.h"
#include "tensorrt_llm/layers/baseBeamSearchLayer.h"
#include "tensorrt_llm/layers/fillBuffers.h"

#include <algorithm>

using namespace tensorrt_llm::common;
using namespace tensorrt_llm::kernels;

namespace tensorrt_llm
{
namespace layers
{

__global__ void update_indir_cache_kernel(int* tgt_indir_cache, const int* src_indir_cache, const int** parent_ids,
    const FinishedState* finished, const int* sequence_lengths, const int* input_lengths, int batch_dim,
    int local_batch_size, int beam_width, int max_attention_window, int sink_token_length, int max_seq_len)
{
    int time_step = threadIdx.x + blockIdx.x * blockDim.x;
    int bb_id = threadIdx.y + blockIdx.y * blockDim.y;   // should be just blockIdx.y?
    const int current_step{sequence_lengths[bb_id] - 1}; // the sequence_lengths is updated, need to minus 1
    const int input_length{input_lengths == nullptr ? 0 : input_lengths[bb_id]};
    const int batch_id = bb_id / beam_width;
    const int beam_id = bb_id % beam_width;
    // Exit when the batch_beam or timestep is out of the bound.
    // Assume that KV Cache is shared and fixed for context part,
    //  so we don't need to update the indices for context part.
    if (bb_id >= beam_width * local_batch_size || time_step >= max_seq_len || time_step < input_length
        || time_step < (max_seq_len - max_attention_window) || finished[bb_id].isFinished())
    {
        return;
    }
    int time_step_circ = time_step;
    if (time_step_circ >= sink_token_length)
    {
        time_step_circ
            = sink_token_length + (time_step - sink_token_length) % (max_attention_window - sink_token_length);
    }

    // for the parent_ids, we will still keep it for all past tokens (i.e. max_seq_len)
    const int src_beam = parent_ids[batch_id][beam_id * max_seq_len + current_step];

    // for the indir tables, we have the cyclic kv cache.
    const uint32_t tgt_offset
        = batch_id * beam_width * max_attention_window + beam_id * max_attention_window + time_step_circ;
    const uint32_t src_offset
        = batch_id * beam_width * max_attention_window + src_beam * max_attention_window + time_step_circ;

    tgt_indir_cache[tgt_offset] = (time_step == current_step) ? beam_id : src_indir_cache[src_offset];
}

void update_indir_cache_kernelLauncher(int* tgt_indir_cache, const int* src_indir_cache, const int** parent_ids,
    const FinishedState* finished, const int* sequence_lengths, const int* input_lengths, int batch_dim,
    int local_batch_size, int beam_width, int max_seq_len, int max_attention_window, int sink_token_length,
    hipStream_t stream)
{
    const dim3 block(32);
    // Update indirections steps [input_length[bb_id], sequence_lengths[bb_id]], included
    const dim3 grid((max_seq_len + block.x - 1) / block.x, local_batch_size * beam_width);
    update_indir_cache_kernel<<<grid, block, 0, stream>>>(tgt_indir_cache, src_indir_cache, parent_ids, finished,
        sequence_lengths, input_lengths, batch_dim, local_batch_size, beam_width, max_attention_window,
        sink_token_length, max_seq_len);
}

template <typename T>
BaseBeamSearchLayer<T>::BaseBeamSearchLayer(
    size_t vocab_size, size_t vocab_size_padded, hipStream_t stream, std::shared_ptr<IAllocator> allocator)
    : BaseLayer(stream, std::move(allocator), nullptr)
    , vocab_size_(vocab_size)
    , vocab_size_padded_(vocab_size_padded)
{
}

template <typename T>
BaseBeamSearchLayer<T>::BaseBeamSearchLayer(BaseBeamSearchLayer<T> const& beam_search_layer)
    : BaseLayer(beam_search_layer)
    , vocab_size_(beam_search_layer.vocab_size_)
    , vocab_size_padded_(beam_search_layer.vocab_size_padded_)
    , topk_softmax_workspace_size_(beam_search_layer.topk_softmax_workspace_size_)
{
}

template <typename T>
BaseBeamSearchLayer<T>::~BaseBeamSearchLayer()
{
    TLLM_LOG_TRACE(__PRETTY_FUNCTION__);
    freeBuffer();
}

template <typename T>
void BaseBeamSearchLayer<T>::freeBuffer()
{
    TLLM_LOG_TRACE("%s start", __PRETTY_FUNCTION__);
    if (mIsAllocateBuffer)
    {
        mAllocator->free((void**) (&temperature_buf_));
        mAllocator->free((void**) (&min_lengths_buf_));
        mAllocator->free((void**) (&repetition_penalty_buf_));
        mAllocator->free((void**) (&presence_penalty_buf_));
        mAllocator->free((void**) (&frequency_penalty_buf_));
        mIsAllocateBuffer = false;
    }
    TLLM_LOG_TRACE("%s stop", __PRETTY_FUNCTION__);
}

template <typename T>
void BaseBeamSearchLayer<T>::allocateBuffer(size_t batch_size)
{
    TLLM_LOG_TRACE("%s start", __PRETTY_FUNCTION__);
    temperature_buf_ = mAllocator->reMalloc(temperature_buf_, sizeof(float) * batch_size, false);
    min_lengths_buf_ = mAllocator->reMalloc(min_lengths_buf_, sizeof(int) * batch_size, false);
    repetition_penalty_buf_ = mAllocator->reMalloc(repetition_penalty_buf_, sizeof(float) * batch_size, false);
    presence_penalty_buf_ = mAllocator->reMalloc(presence_penalty_buf_, sizeof(float) * batch_size, false);
    frequency_penalty_buf_ = mAllocator->reMalloc(frequency_penalty_buf_, sizeof(float) * batch_size, false);

    mIsAllocateBuffer = true;
    TLLM_LOG_TRACE("%s stop", __PRETTY_FUNCTION__);
}

template <typename T>
void BaseBeamSearchLayer<T>::setupBase(size_t batch_size, SetupParams const& setupParams)
{
    allocateBuffer(batch_size);
    TLLM_LOG_TRACE("%s start", __PRETTY_FUNCTION__);
    // Setup penalties.
    FillBuffers const fillBuffers{batch_size, mStream};

    use_temperature_ = static_cast<bool>(setupParams.temperature);
    use_repetition_penalty_ = static_cast<bool>(setupParams.repetition_penalty);
    use_presence_penalty_ = static_cast<bool>(setupParams.presence_penalty);
    use_frequency_penalty_ = static_cast<bool>(setupParams.frequency_penalty);
    use_min_lengths_ = static_cast<bool>(setupParams.min_length);
    if (use_temperature_)
    {
        fillBuffers(setupParams.temperature, getDefaultPenaltyValue(RepetitionPenaltyType::Temperature), mTemperature,
            temperature_buf_, (float*) nullptr, (int*) nullptr);
    }
    if (use_repetition_penalty_)
    {
        fillBuffers(setupParams.repetition_penalty, getDefaultPenaltyValue(RepetitionPenaltyType::Repetition),
            mRepetitionPenalty, repetition_penalty_buf_, (float*) nullptr, (int*) nullptr);
    }
    if (use_presence_penalty_)
    {
        fillBuffers(setupParams.presence_penalty, getDefaultPenaltyValue(RepetitionPenaltyType::Presence),
            mPresencePenalty, presence_penalty_buf_, (float*) nullptr, (int*) nullptr);
    }
    if (use_frequency_penalty_)
    {
        fillBuffers(setupParams.frequency_penalty, getDefaultPenaltyValue(RepetitionPenaltyType::Frequency),
            mFrequencyPenalty, frequency_penalty_buf_, (float*) nullptr, (int*) nullptr);
    }
    if (use_min_lengths_)
    {
        fillBuffers(setupParams.min_length, (int) getDefaultPenaltyValue(RepetitionPenaltyType::MinLength), mMinLengths,
            min_lengths_buf_, (int*) nullptr, (int*) nullptr);
    }
    TLLM_LOG_TRACE("%s stop", __PRETTY_FUNCTION__);
}

template <typename T>
void BaseBeamSearchLayer<T>::forward(BeamSearchOutputParams& outputs, ForwardParams const& params,
    int* penalty_workspace, const int* penalty_workspace_prev)
{
    TLLM_LOG_TRACE("%s", __PRETTY_FUNCTION__);
    Tensor& output_ids_ptr = outputs.output_ids_ptr;

    const auto batch_size = static_cast<std::int32_t>(output_ids_ptr.shape[0]);
    const auto beam_width = static_cast<std::int32_t>(output_ids_ptr.shape[1]);
    const auto max_seq_len = static_cast<std::int32_t>(output_ids_ptr.shape[2]);

    TLLM_CHECK_WITH_INFO(params.ite == 0, "Pipeline Parallelism is not supported yet !");

    const int ite = params.ite;
    auto* const input_lengths = params.input_lengths ? params.input_lengths->template getPtr<const int>() : nullptr;
    int* sequence_length = (outputs.sequence_length) ? outputs.sequence_length->template getPtr<int>() : nullptr;
    Tensor const& logits = params.logits;
    const auto local_batch_size = logits.shape[0];

#define ALL_OF(p_, sz_, dt_, v_) (std::all_of(p_, p_ + sz_, [&](dt_ b) { return b == v_; }))

    const T* embedding_bias = params.embedding_bias ? params.embedding_bias->template getPtr<const T>() : nullptr;
    auto* temperatures = (use_temperature_
                             && !ALL_OF(std::begin(mTemperature) + ite * local_batch_size, local_batch_size, float,
                                 getDefaultPenaltyValue(RepetitionPenaltyType::Temperature)))
        ? temperature_buf_ + ite * local_batch_size
        : nullptr;
    auto* repetition_penalties
        = (use_repetition_penalty_
              && !ALL_OF(std::begin(mRepetitionPenalty) + ite * local_batch_size, local_batch_size, float,
                  getDefaultPenaltyValue(RepetitionPenaltyType::Repetition)))
        ? repetition_penalty_buf_ + ite * local_batch_size
        : nullptr;
    auto* presence_penalties = (use_presence_penalty_
                                   && !ALL_OF(std::begin(mPresencePenalty) + ite * local_batch_size, local_batch_size,
                                       float, getDefaultPenaltyValue(RepetitionPenaltyType::Presence)))
        ? presence_penalty_buf_ + ite * local_batch_size
        : nullptr;
    auto* frequency_penalties = (use_frequency_penalty_
                                    && !ALL_OF(std::begin(mFrequencyPenalty) + ite * local_batch_size, local_batch_size,
                                        float, getDefaultPenaltyValue(RepetitionPenaltyType::Frequency)))
        ? frequency_penalty_buf_ + ite * local_batch_size
        : nullptr;
    auto* min_lengths = (use_min_lengths_
                            && !ALL_OF(std::begin(mMinLengths) + ite * local_batch_size, local_batch_size, int,
                                (int) getDefaultPenaltyValue(RepetitionPenaltyType::MinLength)))
        ? min_lengths_buf_ + ite * local_batch_size
        : nullptr;

    InvokeBatchApplyPenaltyParams<T> penalty_params{logits.getPtr<T>(), embedding_bias,
        penalty_workspace + ite * local_batch_size * beam_width * vocab_size_,
        penalty_workspace_prev + ite * local_batch_size * beam_width * vocab_size_, temperatures, repetition_penalties,
        presence_penalties, frequency_penalties,
        (use_repetition_penalty_ || use_presence_penalty_ || use_frequency_penalty_), local_batch_size, beam_width,
        max_seq_len, vocab_size_, vocab_size_padded_, output_ids_ptr.template getPtr<const int*>(),
        outputs.parent_ids_ptr.template getPtr<const int*>(), input_lengths, sequence_length, min_lengths,
        params.end_ids.template getPtr<const int>(), nullptr, mStream};
    invokeBatchApplyPenalty(penalty_params);
    sync_check_cuda_error();

    invokeSoftMax(outputs, params);
    sync_check_cuda_error();

    if (beam_width > 1)
    {
        update_indir_cache_kernelLauncher(outputs.tgt_cache_indirection.template getPtr<int>(),
            params.src_cache_indirection.template getPtr<const int>(),
            outputs.parent_ids_ptr.template getPtr<const int*>(),
            reinterpret_cast<const FinishedState*>(
                outputs.finished->template getPtr<const FinishedState::UnderlyingType>()),
            sequence_length, input_lengths, batch_size, local_batch_size, beam_width, max_seq_len,
            params.max_attention_window, params.sink_token_length, mStream);
        sync_check_cuda_error();
    }
}

template class BaseBeamSearchLayer<float>;
template class BaseBeamSearchLayer<half>;

} // namespace layers
} // namespace tensorrt_llm
