#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2024, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "tensorrt_llm/common/memoryUtils.h"
#include "tensorrt_llm/kernels/beamSearchKernels.h"
#include "tensorrt_llm/layers/beamSearchLayer.h"
#include "tensorrt_llm/layers/defaultDecodingParams.h"
#include "tensorrt_llm/layers/layerUtils.h"
#include <limits>

using namespace tensorrt_llm::common;
using namespace tensorrt_llm::kernels;

namespace tensorrt_llm
{
namespace layers
{

template <typename T>
BeamSearchLayer<T>::BeamSearchLayer(
    DecoderDomain const& decoderDomain, hipStream_t stream, std::shared_ptr<IAllocator> allocator)
    : BaseLayer(decoderDomain, stream, std::move(allocator))
    , mVocabSize(decoderDomain.getVocabSize())
    , mVocabSizePadded(decoderDomain.getVocabSizePadded())
{
    TLLM_LOG_TRACE(__PRETTY_FUNCTION__);
}

template <typename T>
BeamSearchLayer<T>::~BeamSearchLayer()
{
    TLLM_LOG_TRACE(__PRETTY_FUNCTION__);
}

template <typename T>
void BeamSearchLayer<T>::setup(runtime::SizeType32 const batchSize, runtime::SizeType32 const beamWidth,
    runtime::SizeType32 const* batchSlots, std::shared_ptr<BaseSetupParams> baseSetupParams)
{
    TLLM_LOG_TRACE("%s start", __PRETTY_FUNCTION__);
    TLLM_CHECK_WITH_INFO(
        beamWidth <= nMaxBeamWidth, std::string("Beam width is larger than the maximum supported (64)."));

    auto setupParams = std::dynamic_pointer_cast<BeamSearchSetupParams>(baseSetupParams);

    mDiversityRateHost.resize(batchSize);
    mLengthPenaltyHost.resize(batchSize);
    mEarlyStoppingHost.resize(batchSize);
    allocateBuffer(batchSize, beamWidth);

    auto constexpr fltMax = std::numeric_limits<float>::max();
    auto constexpr fltMin = std::numeric_limits<float>::lowest();
    auto constexpr fltEpsilon = std::numeric_limits<float>::epsilon();

    FillBuffers const fillBuffers{batchSize, batchSize, mStream};
    fillBuffers(setupParams->beam_search_diversity_rate, DefaultDecodingParams::getBeamSearchDiversity(),
        mDiversityRateHost, mDiversityRateDevice, (int*) nullptr, std::make_pair(-fltEpsilon, fltMax),
        "diveristy rate");
    fillBuffers(setupParams->length_penalty, DefaultDecodingParams::getLengthPenalty(), mLengthPenaltyHost,
        mLengthPenaltyDevice, (int*) nullptr, std::make_pair(fltMin, fltMax), "length penalty");
    fillBuffers(setupParams->early_stopping, DefaultDecodingParams::getEarlyStopping(), mEarlyStoppingHost,
        mEarlyStoppingDevice, (int*) nullptr, std::make_pair(fltMin, fltMax), "early stopping");
    mHasDiffRuntimeArgs = setupParams->hasDiffRuntimeArgs;

    TLLM_LOG_TRACE("%s stop", __PRETTY_FUNCTION__);
}

__global__ void updateCacheIndirectionKernel(
    int* tgtCI, int const* srcCI, BeamHypotheses bh, int const nMaxAttentionWindow, int const nSinkTokenLength)
{
    // Update indirections from steps `bh.inputLength[indexBatchBeam]` to step `sequence_lengths[indexBatchBeam]`
    int const step = threadIdx.x + blockIdx.x * blockDim.x;
    int const indexBatchBeam = blockIdx.y;
    int const nBS{bh.nBatchSize};
    int const nBM{bh.nBeamWidth};
    int const nMSL{bh.nMaxSeqLen};
    int const indexBatch = indexBatchBeam / nBM;
    int const indexBeam = indexBatchBeam % nBM;
    int const lastStep{bh.sequenceLengths[indexBatchBeam] - 1}; // the sequence_lengths is updated, need to minus 1

    // Return early when the indexBatchBeam or step is out of the bound
    // No update for the indices of context part since KV Cache is shared
    if (indexBatchBeam >= nBM * nBS || step >= nMSL || step < bh.inputLengths[indexBatchBeam]
        || step < (nMSL - nMaxAttentionWindow) || bh.finished[indexBatchBeam].isFinished())
    {
        return;
    }

    // Keep all past tokens by parentIdsPtr
    int const indexBeamSrc = bh.parentIdsPtr[indexBatch][indexBeam * nMSL + lastStep];
    int const stepCirc = (step >= nSinkTokenLength)
        ? nSinkTokenLength + (step - nSinkTokenLength) % (nMaxAttentionWindow - nSinkTokenLength)
        : step;
    // Consider cyclic kv cache for the indir tables
    uint32_t const tgtOffset = indexBatch * nBM * nMaxAttentionWindow + indexBeam * nMaxAttentionWindow + stepCirc;
    uint32_t const srcOffset = indexBatch * nBM * nMaxAttentionWindow + indexBeamSrc * nMaxAttentionWindow + stepCirc;
    tgtCI[tgtOffset] = (step == lastStep) ? indexBeam : srcCI[srcOffset];
}

template <typename T>
void BeamSearchLayer<T>::forwardAsyncSingleRequest(
    std::shared_ptr<BaseOutputParams> baseOutputs, std::shared_ptr<BaseInputParams> baseInputs)
{
    TLLM_LOG_TRACE("%s start", __PRETTY_FUNCTION__);

    auto ip = std::dynamic_pointer_cast<BeamSearchInputParams>(baseInputs);
    auto op = std::dynamic_pointer_cast<BeamSearchOutputParams>(baseOutputs);

    TLLM_CHECK_WITH_INFO(op->beamHypotheses, std::string("Output BeamHypotheses is not set."));
    TLLM_CHECK_WITH_INFO(op->sequence_length->template getPtr<int>() != nullptr || mLengthPenaltyDevice == nullptr,
        std::string("Current sequence lengths must be set for length penalty computation."));
    TLLM_CHECK_WITH_INFO(ip->ite == 0, "Pipeline Parallelism is not supported yet !");

    BeamHypotheses& bh{*op->beamHypotheses};
    // bh's members already initialized in op: *CBA, batchDones
    // bh's members not used in function: outputIds, logProbs, outputIdsUnfinish, parentIdsUnfinish
    bh.nMaxBatchSize = static_cast<std::int32_t>(op->output_ids_ptr.shape[0]);
    bh.nBatchSize = ip->logits.shape[0];
    bh.nBeamWidth = static_cast<std::int32_t>(op->output_ids_ptr.shape[1]);
    bh.nIte = ip->ite;
    bh.nMaxSeqLen = static_cast<std::int32_t>(op->output_ids_ptr.shape[2]);
    bh.nVocabSize = mVocabSizePadded;
    bh.diversityRates = mDiversityRateDevice;
    bh.lengthPenalties = mLengthPenaltyDevice;
    bh.earlyStoppings = mEarlyStoppingDevice;
    bh.inputLengths = ip->input_lengths->template getPtr<int const>();
    bh.endIds = ip->end_ids.template getPtr<int const>();
    bh.logProbsTiled = (op->output_log_probs) ? op->output_log_probs->template getPtr<float>() : nullptr;
    bh.sequenceLengths = op->sequence_length->template getPtr<int>();
    bh.cumLogProbs = op->cum_log_probs->template getPtr<float>();
    bh.finished = reinterpret_cast<FinishedState*>(op->finished->template getPtr<FinishedState::UnderlyingType>());
    bh.outputIdsPtr = op->output_ids_ptr.template getPtr<int*>();
    bh.parentIdsPtr = op->parent_ids_ptr.template getPtr<int*>();

    T const* logits = ip->logits.template getPtr<T>();
    T const* bias = static_cast<T const*>(nullptr);
    TLLM_CHECK_WITH_INFO(mWorkspaceSize >= 2 * bh.nBatchSize * bh.nBeamWidth * bh.nBeamWidth * 2,
        fmtstr("Workspace size (%lu) is not enough for topk softmax required (%lu).", (uint64_t) mWorkspaceSize,
            (uint64_t) (2 * bh.nMaxBatchSize * bh.nBeamWidth * bh.nBeamWidth * 2)));

    invokeTopkSoftMax(logits, bias, mWorkspace, bh, mStream);
    sync_check_cuda_error();

    if (bh.nBeamWidth > 1)
    {
        auto tgtCI = op->tgt_cache_indirection.template getPtr<int>();
        auto srcCI = ip->src_cache_indirection.template getPtr<int const>();
        dim3 const grid(roundUp(bh.nMaxSeqLen, 32), bh.nBatchSize * bh.nBeamWidth);
        updateCacheIndirectionKernel<<<grid, 32, 0, mStream>>>(
            tgtCI, srcCI, bh, ip->max_attention_window, ip->sink_token_length);
        sync_check_cuda_error();
    }

    TLLM_LOG_TRACE("%s stop", __PRETTY_FUNCTION__);
}

template <typename T>
void BeamSearchLayer<T>::forwardAsync(
    std::shared_ptr<BaseOutputParams> baseOutputs, std::shared_ptr<BaseInputParams> baseInputs)
{
    TLLM_LOG_TRACE("%s start", __PRETTY_FUNCTION__);
    auto outputs = std::dynamic_pointer_cast<DynamicDecodeOutputParams>(baseOutputs);
    auto params = std::dynamic_pointer_cast<DynamicDecodeInputParams>(baseInputs);

    auto const localDecoderDomain = getLocalDecoderDomain(params, mDecoderDomain);

    auto batchSlots = params->batch_slots ? params->batch_slots->template getPtr<SizeType32 const>() : nullptr;
    auto const maxSeqLen = outputs->output_ids.shape[outputs->output_ids.shape.size() - 1];
    auto const ite = params->ite;
    auto const step = params->step;

    // common inputs
    auto const& endIds = params->end_ids;
    auto const localBatchSize = static_cast<std::size_t>(params->local_batch_size);

    TLLM_CHECK_WITH_INFO(localDecoderDomain.getBeamWidth() > 1,
        "Decoding mode is beam search, but beamWidth <= 1 (%d <= 1)", localDecoderDomain.getBeamWidth());
    TLLM_CHECK_WITH_INFO(
        params->src_cache_indirection.has_value(), "src_cache_indirection is mandatory in beam search.");
    TLLM_CHECK_WITH_INFO(
        outputs->tgt_cache_indirection.has_value(), "tgt_cache_indirection is mandatory in beam search.");
    TLLM_CHECK_WITH_INFO(outputs->parent_ids.has_value(), "parent_ids tensor is mandatory in beam search.");
    TLLM_CHECK_WITH_INFO(outputs->finished.has_value(), "finished tensor is mandatory in beam search.");
    TLLM_CHECK_WITH_INFO(outputs->cum_log_probs.has_value(), "cum_log_probs tensor is mandatory in beam search.");

    // Compute one by one if there are different runtime arguments
    //     due to Batch-Beam-Search is not supported yet, so we need to compute
    size_t const dynamic_decode_batch_size = mHasDiffRuntimeArgs ? 1 : localBatchSize;
    auto const dynamic_decode_total_iteration = mHasDiffRuntimeArgs ? localBatchSize : 1;

    for (uint32_t dynamic_ite = 0; dynamic_ite < dynamic_decode_total_iteration; ++dynamic_ite)
    {
        auto const dynamic_id_offset = dynamic_ite * dynamic_decode_batch_size * localDecoderDomain.getBeamWidth();
        auto const dynamic_decode_vocab_size_units_offset = dynamic_id_offset * mDecoderDomain.getVocabSizePadded();

        auto const logits_offset
            = params->logits->slice({dynamic_decode_batch_size, params->logits->shape[1], params->logits->shape[2]},
                dynamic_decode_vocab_size_units_offset);
        auto const end_id_offset = endIds.slice({dynamic_decode_batch_size}, dynamic_ite * dynamic_decode_batch_size);

        auto forwardParams = std::make_shared<BeamSearchInputParams>(step, ite, logits_offset, end_id_offset,
            *params->src_cache_indirection, static_cast<std::int32_t>(params->max_attention_window),
            static_cast<std::int32_t>(params->sink_token_length), static_cast<std::int32_t>(maxSeqLen));

        if (params->input_lengths)
        {
            forwardParams->input_lengths = params->input_lengths->slice(
                {dynamic_decode_batch_size * localDecoderDomain.getBeamWidth()}, dynamic_id_offset);
        }

        auto outputParams = std::make_shared<BeamSearchOutputParams>(
            outputs->output_ids, outputs->parent_ids.value(), outputs->tgt_cache_indirection.value());

        outputParams->output_ids_ptr = std::move(outputs->output_ids_ptr);
        outputParams->parent_ids_ptr = std::move(outputs->parent_ids_ptr);
        outputParams->sequence_length = outputs->sequence_length->slice(
            {dynamic_decode_batch_size * localDecoderDomain.getBeamWidth()}, dynamic_id_offset);
        outputParams->finished = outputs->finished->slice(
            {dynamic_decode_batch_size * localDecoderDomain.getBeamWidth()}, dynamic_id_offset);
        outputParams->cum_log_probs = outputs->cum_log_probs->slice(
            {dynamic_decode_batch_size * localDecoderDomain.getBeamWidth()}, dynamic_id_offset);
        outputParams->output_log_probs = outputs->output_log_probs_tiled; // notice: use tiled tensor
        outputParams->beamHypotheses = std::move(outputs->beamHypotheses);

        // beam_search_diversity_rate is only supported when using BeamHypotheses
        forwardAsyncSingleRequest(outputParams, forwardParams);
    } // end of dynamic_ite
    TLLM_LOG_TRACE("%s stop", __PRETTY_FUNCTION__);
}

template <typename T>
void BeamSearchLayer<T>::allocateBuffer(runtime::SizeType32 const batchSize, runtime::SizeType32 const beamWidth)
{
    TLLM_LOG_TRACE("%s start", __PRETTY_FUNCTION__);
    int const nPadBeamWidth = padToNextPowerOfTwo(beamWidth);
    // Unit of mWorkspaceSize is number of elements (not Byte), align to 4 for further optimization
    size_t nTopK = batchSize * nPadBeamWidth * nPadBeamWidth * 2;
    size_t nTempBuffer = batchSize * nPadBeamWidth * nMaxVocabPartForStage1FastKernel * (2 * (nPadBeamWidth * 2) + 2);
    mWorkspaceSize = roundUp(nTopK, 4) * 2 + roundUp(nTempBuffer, 4);
    mWorkspace = mAllocator->reMalloc(mWorkspace, sizeof(float) * mWorkspaceSize, true);
    mDiversityRateDevice = mAllocator->reMalloc(mDiversityRateDevice, sizeof(float) * batchSize, false);
    mLengthPenaltyDevice = mAllocator->reMalloc(mLengthPenaltyDevice, sizeof(float) * batchSize, false);
    mEarlyStoppingDevice = mAllocator->reMalloc(mEarlyStoppingDevice, sizeof(int) * batchSize, false);
    mIsAllocateBuffer = true;
    TLLM_LOG_TRACE("%s stop", __PRETTY_FUNCTION__);
}

template <typename T>
void BeamSearchLayer<T>::freeBuffer()
{
    TLLM_LOG_TRACE("%s start", __PRETTY_FUNCTION__);
    if (mIsAllocateBuffer)
    {
        mAllocator->free((void**) (&mWorkspace));
        mAllocator->free((void**) (&mDiversityRateDevice));
        mAllocator->free((void**) (&mLengthPenaltyDevice));
        mAllocator->free((void**) (&mEarlyStoppingDevice));
        mIsAllocateBuffer = false;
    }
    TLLM_LOG_TRACE("%s stop", __PRETTY_FUNCTION__);
}

template class BeamSearchLayer<float>;
template class BeamSearchLayer<half>;

} // namespace layers
} // namespace tensorrt_llm
